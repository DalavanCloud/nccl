#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENCE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void print_header() {
  PRINT("# %10s  %12s  %6s  %6s        out-of-place                    in-place\n", "", "", "", "");
  PRINT("# %10s  %12s  %6s  %6s  %7s  %5s  %5s  %7s  %7s  %5s  %5s  %7s\n", "bytes", "N", "type", "op",
      "time", "algbw", "busbw", "res", "time", "algbw", "busbw", "res");
}

void print_line_header (int size, int count, const char *typeName, const char *opName, int root) {
  PRINT("%12i  %12i  %6s  %6s", size, count, typeName, opName);
}

void getCollByteCount(size_t *sendbytes, size_t *recvbytes, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t *procSharedBytes, int *sameExpected, size_t nbytes, int nranks) {
    *sendbytes = nbytes;
    *recvbytes = nbytes;
    *sameExpected = 1;
    *procSharedBytes = 0;
    *sendInplaceOffset = 0;
    *recvInplaceOffset = 0;
 }

void InitRecvResult(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place, int is_first) {
  size_t count = args->nbytes / wordSize(type);

  while (args->sync[0] != args->thread) pthread_yield();

  for (int i=0; i<args->nGpus; i++) {
    int device;
    NCCLCHECK(ncclCommCuDevice(args->comms[i], &device));
    CUDACHECK(hipSetDevice(device));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];

    if (is_first && i == 0) {
      CUDACHECK(hipMemcpy(args->expected[0], data, count*wordSize(type), hipMemcpyDeviceToHost));
    } else {
      Accumulate(args->expected[0], data, count, type, op);
    }

    if (in_place == 0) {
      CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->nbytes));
    }
    CUDACHECK(hipDeviceSynchronize());
  }

  args->sync[0] = args->thread + 1;

  if (args->thread+1 == args->nThreads) {
#ifdef MPI_SUPPORT
    // Last thread does the MPI reduction
    if (args->nbytes > 0) {
      void* remote, *remoteHost = malloc(args->nbytes);
      void* myInitialData = malloc(args->nbytes);
      memcpy(myInitialData, args->expectedHost[0], args->nbytes);
      CUDACHECK(hipHostRegister(remoteHost, args->nbytes, 0));
      CUDACHECK(hipHostGetDevicePointer(&remote, remoteHost, 0));
      for (int i=0; i<args->nProcs; i++) {
        if (i == args->proc) {
          MPI_Bcast(myInitialData, args->nbytes, MPI_BYTE, i, MPI_COMM_WORLD);
          free(myInitialData);
        } else {
          MPI_Bcast(remoteHost, args->nbytes, MPI_BYTE, i, MPI_COMM_WORLD);
          Accumulate(args->expected[0], remote, count, type, op);
          hipDeviceSynchronize();
        }
      }
      CUDACHECK(hipHostUnregister(remoteHost));
      free(remoteHost);
    }
#endif
    args->sync[0] = 0;
  } else {
    while (args->sync[0]) pthread_yield();
  }
}

void GetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(2*(nranks - 1)))/((double)nranks);
  *busBw = baseBw * factor;
}

void RunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  NCCLCHECK(ncclAllReduce(sendbuff, recvbuff, count, type, op, comm, stream));
}

void RunTestOp(struct threadArgs_t* args, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  TimeTest(args, type, typeName, op, opName, 0, 1);
}

void RunTestType(struct threadArgs_t* args, ncclDataType_t type, const char* typeName) {
  RunTestOp(args, type, typeName, ncclSum, "sum");
  RunTestOp(args, type, typeName, ncclProd, "prod");
  RunTestOp(args, type, typeName, ncclMax, "max");
  RunTestOp(args, type, typeName, ncclMin, "min");
}

void RunTests(struct threadArgs_t* args) {
  RunTestType(args, ncclInt8, "int8");
  RunTestType(args, ncclUint8, "uint8");
  RunTestType(args, ncclInt32, "int32");
  RunTestType(args, ncclUint32, "uint32");
  RunTestType(args, ncclInt64, "int64");
  RunTestType(args, ncclUint64, "uint64");
  RunTestType(args, ncclHalf, "half");
  RunTestType(args, ncclFloat, "float");
  RunTestType(args, ncclDouble, "double");
}
