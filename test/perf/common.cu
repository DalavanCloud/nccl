#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENCE.txt for license information
 ************************************************************************/

#include "common.h"
#include <pthread.h>
#include <cstdio>
#include <getopt.h>

#ifdef MPI_TRANSPORT
extern "C" {
void ncclMpiHook(MPI_Comm comm);
void ncclMpiLock();
void ncclMpiUnlock();
}

#define MPI_PROTECT(mpicall) do { \
  ncclMpiLock(); \
  mpicall; \
  ncclMpiUnlock(); \
} while(0)
#else
#define MPI_PROTECT(mpicall) mpicall
#endif

ncclDataType_t test_types[9] = {ncclInt8, ncclUint8, ncclInt32, ncclUint32, ncclInt64, ncclUint64, ncclHalf, ncclFloat, ncclDouble};
const char *test_typenames[9] = {"int8", "uint8", "int32", "uint32", "int64", "uint64", "half", "float", "double"};
ncclRedOp_t test_ops[4] = {ncclSum, ncclProd, ncclMax, ncclMin};
const char *test_opnames[4] = {"sum", "prod", "max", "min"};

thread_local int is_main_thread = 0;

static int datacheck = 1;
static int iters = 20;
static int ncclop = ncclSum;
static char ncclopstring[10] = "sum";
static int nccltype = ncclFloat;
static char nccltypestring[10] = "float";
static int ncclroot = 0;

double parsesize(char *value) {
    long long int units;
    double size;

    if (strchr(value, 'G') != NULL) {
        units=1024*1024*1024;
    } else if (strchr(value, 'M') != NULL) {
        units=1024*1024;
    } else if (strchr(value, 'K') != NULL) {
        units=1024;
    } else {
        units=1;
    }

    size = atof(value)*units;
    return size;
}

double DeltaMaxValue(ncclDataType_t type) {
  switch(type) {
    case ncclHalf: return 5e-2;
    case ncclFloat: return 1e-5;
    case ncclDouble: return 1e-12;
    case ncclInt8:
    case ncclUint8:
    case ncclInt32:
    case ncclUint32:
    case ncclInt64:
    case ncclUint64: return 1e-200;
  }
  return 1e-200;
}

template<typename T> __device__
double absDiff(T a, T b) {
  return fabs((double)(b - a));
}

template<> __device__
double absDiff<half>(half a, half b) {
  float x = __half2float(a);
  float y = __half2float(b);
  return fabs((double)(y-x));
}

template<typename T> __device__
float toFloat(T a) {
  return (float)a;
}
template<> __device__ 
float toFloat(half a) {
  return __half2float(a);
}


template<typename T, int BSIZE> __global__
void deltaKern(void* A_, void* B_, int count, double* max) {
  const T* A = (const T*)A_;
  const T* B = (const T*)B_;
  __shared__ double temp[BSIZE];
  int tid = threadIdx.x;
  double locmax = 0.0;
  for(int i=tid; i<count; i+=blockDim.x) {

    double delta = absDiff(A[i], B[i]);
    if( delta > locmax ) {
      locmax = delta;
#ifdef DEBUG_PRINT
      if (delta > .1) printf("Error at %d/%d : %f != %f\n", i, count, toFloat(A[i]), toFloat(B[i]));
#endif
    }
  }

  temp[tid] = locmax;
  for(int stride = BSIZE/2; stride > 1; stride>>=1) {
    __syncthreads();
    if( tid < stride )
      temp[tid] = temp[tid] > temp[tid+stride] ? temp[tid] : temp[tid+stride];
  }
  __syncthreads();
  if( threadIdx.x == 0)
    *max = temp[0] > temp[1] ? temp[0] : temp[1];
}


void CheckDelta(void* expected, void* results, int count, ncclDataType_t type, double* devmax) {
  switch (type) {
    case ncclHalf:
      deltaKern<half, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclFloat:
      deltaKern<float, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclDouble:
      deltaKern<double, 512><<<1, 512>>>(results, expected, count, devmax); break;

    case ncclInt8:
    case ncclUint8:
      deltaKern<uint8_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclInt32:
    case ncclUint32:
      deltaKern<uint32_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
    case ncclInt64:
    case ncclUint64:
      deltaKern<uint64_t, 512><<<1, 512>>>(results, expected, count, devmax); break;
  }
}

#define CURAND_CHK(cmd)                                                         \
    do {                                                                        \
      hiprandStatus_t error = (cmd);                                             \
      if (error != HIPRAND_STATUS_SUCCESS) {                                     \
        printf("CuRAND error %i at %s:%i\n", error, __FILE__ , __LINE__);       \
        exit(EXIT_FAILURE);                                                     \
      }                                                                         \
    } while (false)


template<typename T>
void GenerateRandom(hiprandGenerator_t generator, T * const dest,
    const int N);

template<>
void GenerateRandom<int8_t>(hiprandGenerator_t generator, int8_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest,
      N * sizeof(int8_t) / sizeof(int)));
}
template<>
void GenerateRandom<uint8_t>(hiprandGenerator_t generator, uint8_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest,
      N * sizeof(uint8_t) / sizeof(int)));
}

template<>
void GenerateRandom<int32_t>(hiprandGenerator_t generator, int32_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest, N));
}

template<>
void GenerateRandom<uint32_t>(hiprandGenerator_t generator, uint32_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int*)dest, N));
}

template<>
void GenerateRandom<float>(hiprandGenerator_t generator, float * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerateUniform(generator, dest, N));
}

template<>
void GenerateRandom<double>(hiprandGenerator_t generator, double * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerateUniformDouble(generator, dest, N));
}

template<>
void GenerateRandom<uint64_t>(hiprandGenerator_t generator, uint64_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int *)dest, N*2));
}

template<>
void GenerateRandom<int64_t>(hiprandGenerator_t generator, int64_t * const dest,
    const int N) {
  CURAND_CHK(hiprandGenerate(generator, (unsigned int *)dest, N*2));
}

template<typename T>
void RandomizeType(void* dest, const int N, const int randomSeed) {
  T* ptr = (T*)dest;
  hiprandGenerator_t gen;
  CURAND_CHK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
  CURAND_CHK(hiprandSetPseudoRandomGeneratorSeed(gen, randomSeed));
  GenerateRandom<T>(gen, ptr, N);
  CURAND_CHK(hiprandDestroyGenerator(gen));
  CUDACHECK(hipDeviceSynchronize());
}

__global__ void halve(const float * src, half* dest, int N) {
  for(int tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < N; tid += blockDim.x * gridDim.x)
    dest[tid] = __float2half(src[tid]);
}

void RandomizeHalf(void* dest, const int N, const int randomSeed) {
  half* ptr = (half*)dest;
  hiprandGenerator_t gen;
  CURAND_CHK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32));
  CURAND_CHK(hiprandSetPseudoRandomGeneratorSeed(gen, randomSeed));

  float* temp;
  CUDACHECK(hipMalloc(&temp, N*sizeof(float)));
  GenerateRandom<float>(gen, temp, N);
  halve<<<128, 512>>>(temp, ptr, N);
  CURAND_CHK(hiprandDestroyGenerator(gen));
  CUDACHECK(hipFree(temp));
  CUDACHECK(hipDeviceSynchronize());
}

void Randomize(void* ptr, const int count, ncclDataType_t type, const int seed) {
  switch (type) {
    case ncclInt8:   RandomizeType<int8_t>  (ptr, count, seed); break;
    case ncclUint8:  RandomizeType<uint8_t> (ptr, count, seed); break;
    case ncclInt32:  RandomizeType<int32_t> (ptr, count, seed); break;
    case ncclUint32: RandomizeType<uint32_t>(ptr, count, seed); break;
    case ncclInt64:  RandomizeType<int64_t> (ptr, count, seed); break;
    case ncclUint64: RandomizeType<uint64_t>(ptr, count, seed); break;
    case ncclHalf:   RandomizeHalf          (ptr, count, seed); break;
    case ncclFloat:  RandomizeType<float>   (ptr, count, seed); break;
    case ncclDouble: RandomizeType<double>  (ptr, count, seed); break;
  }
}

template<typename T, int OP> __global__ static
void accumKern(T* acum, const T* contrib, int N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    T c = contrib[i];
    T a = acum[i];
    if(OP == ncclSum) {
      acum[i] = a+c;
    } else if(OP == ncclProd) {
      acum[i] = a*c;
    } else if(OP == ncclMax) {
      acum[i] = (a > c) ? a : c;
    } else if(OP == ncclMin) {
      acum[i] = (a < c) ? a : c;
    }
  }
}

template<> __global__
void accumKern<half, ncclSum>(half* acum, const half* contrib, int N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( a + c );
  }
}

template<> __global__
void accumKern<half, ncclProd>(half* acum, const half* contrib, int N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( a * c );
  }
}

template<> __global__
void accumKern<half, ncclMax>(half* acum, const half* contrib, int N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( (a>c) ? a : c );
  }
}

template<> __global__
void accumKern<half, ncclMin>(half* acum, const half* contrib, int N) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int offset = blockDim.x*gridDim.x;
  for(int i=tid; i<N; i+=offset) {
    float c = __half2float(contrib[i]);
    float a = __half2float(acum[i]);
    acum[i] = __float2half( (a<c) ? a : c );
  }
}

template<typename T>
void accVecType(void* out, void* in, int n, ncclRedOp_t op) {
  switch(op) {
    case ncclSum:  accumKern<T, ncclSum> <<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclProd: accumKern<T, ncclProd><<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclMax:  accumKern<T, ncclMax> <<<256,256>>>((T*)out, (T*)in, n); break;
    case ncclMin:  accumKern<T, ncclMin> <<<256,256>>>((T*)out, (T*)in, n); break;
    default:
      printf("Unknown reduction operation.\n");
      exit(EXIT_FAILURE);
  }
}

void Accumulate(void* out, void* in, int n, ncclDataType_t type, ncclRedOp_t op) {
  switch (type) {
    case ncclInt8:   accVecType<int8_t>   (out, in, n, op); break;
    case ncclUint8:  accVecType<uint8_t>  (out, in, n, op); break;
    case ncclInt32:  accVecType<int32_t>  (out, in, n, op); break;
    case ncclUint32: accVecType<uint32_t> (out, in, n, op); break;
    case ncclInt64:  accVecType<int64_t>  (out, in, n, op); break;
    case ncclUint64: accVecType<uint64_t> (out, in, n, op); break;
    case ncclHalf:   accVecType<half>     (out, in, n, op); break;
    case ncclFloat:  accVecType<float>    (out, in, n, op); break;
    case ncclDouble: accVecType<double>   (out, in, n, op); break;
    default:
      printf("Unknown reduction type.\n");
      exit(EXIT_FAILURE);
  }
}

#ifdef MPI_TRANSPORT
extern "C"
void ncclMpiHook(MPI_Comm comm);
#endif

void Barrier(struct threadArgs_t* args)
{
  static int i = 0;

  while (args->sync[i] != args->thread) pthread_yield();
  args->sync[i] = args->thread + 1;
  if (args->thread+1 == args->nThreads) {
#ifdef MPI_SUPPORT
    MPI_Barrier(MPI_COMM_WORLD);
#endif
    args->sync[i] = 0;
  } else {
    while (args->sync[i]) pthread_yield();
  }

  i=!i;
}

void RandomizeAccumulate(void* data, void* accum, int count, ncclDataType_t type, ncclRedOp_t op, int seed, int rank) {
  Randomize(data, count, type, seed);
  if (rank == 0) {
    CUDACHECK(hipMemcpy(accum, data, count*wordSize(type), hipMemcpyDeviceToHost));
  } else {
    Accumulate(accum, data, count, type, op);
  }
}

double CheckData(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place) {
  int count = args->expectedBytes/wordSize(type);
  double maxDelta = 0.0;
  for (int i=0; i<args->nGpus; i++) {
    int device;
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    NCCLCHECK(ncclCommCuDevice(args->comms[i], &device));
    CUDACHECK(hipSetDevice(device));
    void *data = in_place ? ((void *)((uintptr_t)args->recvbuffs[i] + args->recvInplaceOffset*rank)) : args->recvbuffs[i];
    CheckDelta(data , args->expected[i], count, type, args->delta);
    hipDeviceSynchronize();
    maxDelta = std::max(*(args->deltaHost), maxDelta);

#ifdef DEBUG_PRINT
    if (rank == 0) { 
       int *temp = (int *)malloc(args->expectedBytes);

       printf("\n Expected: ");
       for(int j=0; j<args->expectedBytes/sizeof(int); j++) { 
       	printf("%d:%d ", j, *((int *)args->expectedHost[0] + j));
       }
       printf("\n");

       hipMemcpy(temp, data, args->expectedBytes, hipMemcpyDeviceToHost);
       printf("\n Actual: ");
       for (int j=0; j<args->expectedBytes/sizeof(int); j++) { 
       	printf("%d:%d ", j, *((int *)temp + j));
       }
       printf("\n");
       free(temp);
    }
#endif
  }
  if (maxDelta > DeltaMaxValue(type)) args->errors[0]++;
  return maxDelta;
}

void InitSend(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place, int is_first) {
  size_t count = args->sendBytes / wordSize(type);
  static int rep = 1;
  for (int i=0; i<args->nGpus; i++) {
    int device;
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    NCCLCHECK(ncclCommCuDevice(args->comms[i], &device));
    CUDACHECK(hipSetDevice(device));
    void* data = in_place ? (void *)((uintptr_t)args->recvbuffs[i] + args->sendInplaceOffset*rank) : args->sendbuffs[i];
    int seed = rank+count+rep+in_place;
    Randomize(data, count, type, seed);

#ifdef DEBUG_PRINT
    if (rank == 2) { 
       int *temp = (int *)malloc(args->sendBytes);
       hipMemcpy(temp, data, args->sendBytes, hipMemcpyDeviceToHost);
       printf("\n Send Data at rank %d:", rank);
       for (int i=0; i<args->sendBytes/sizeof(int); i++) { 
       	printf("%d:%d ", i, *((int *)temp + i));
       }
       printf("\n");
    }
#endif

    hipDeviceSynchronize();
  }
  rep++;
}

#define CHECK 1

void BenchTime(struct threadArgs_t* args, ncclDataType_t type, ncclRedOp_t op, int root, int in_place) {
  size_t count = args->nbytes / wordSize(type);
  
  // Warmup / Sync
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < args->nGpus; i++) {
    // Intialize data after warmup so that we can overwrite safely
    RunColl((void *)args->sendbuffs[i], 
        (void *)args->recvbuffs[i], count, type, op, root, args->comms[i], args->streams[i]);
  }
  NCCLCHECK(ncclGroupEnd());
  for (int i = 0; i < args->nGpus; ++i) {
    CUDACHECK(hipStreamSynchronize(args->streams[i]));
  }

  Barrier(args);

  // Performance Benchmark
  auto start = std::chrono::high_resolution_clock::now();
  for (int iter = 0; iter < iters; iter++) { 
      NCCLCHECK(ncclGroupStart());
      for (int i = 0; i < args->nGpus; i++) {
        int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
        RunColl((void*)(in_place ? ((void *)((uintptr_t)args->recvbuffs[i] + args->sendInplaceOffset*rank)) : args->sendbuffs[i]), 
            (void*)(in_place ? (void*)((uintptr_t)args->recvbuffs[i] + args->recvInplaceOffset*rank) : args->recvbuffs[i]), 
            count, type, op, root, args->comms[i], args->streams[i]);
      }
      NCCLCHECK(ncclGroupEnd());
  }

  for (int i = 0; i < args->nGpus; ++i) {
    CUDACHECK(hipStreamSynchronize(args->streams[i]));
  }
  auto delta = std::chrono::high_resolution_clock::now() - start;
  double deltaSec = std::chrono::duration_cast<std::chrono::duration<double>>(delta).count();
  deltaSec = deltaSec/iters;

  double algBw, busBw;
  GetBw(count, wordSize(type), deltaSec, &algBw, &busBw, args->nProcs*args->nThreads*args->nGpus);

  Barrier(args);

  if (datacheck) { 
      InitSend(args, type, op, root, in_place, args->thread == 0 ? 1 : 0);
      InitRecvResult(args, type, op, root, in_place, args->thread == 0 ? 1 : 0);
      hipDeviceSynchronize();
  }

  //test validation in single itertion, should ideally be included into the multi-iteration run
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < args->nGpus; i++) {
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    RunColl((void*)(in_place ? ((void *)((uintptr_t)args->recvbuffs[i] + args->sendInplaceOffset*rank)) : args->sendbuffs[i]), 
        (void*)(in_place ? (void*)((uintptr_t)args->recvbuffs[i] + args->recvInplaceOffset*rank) : args->recvbuffs[i]), 
        count, type, op, root, args->comms[i], args->streams[i]);
  }
  NCCLCHECK(ncclGroupEnd());

  for (int i = 0; i < args->nGpus; ++i) {
    CUDACHECK(hipStreamSynchronize(args->streams[i]));
  }

  double maxDelta = 0;
#ifdef CHECK
  if (datacheck) { 
     maxDelta = CheckData(args, type, op, root, in_place);
  } else { 
     maxDelta = -1.0;
  }
#else
     maxDelta = -1.0;
#endif

  //aggregate delta from all threads and procs
  Barrier(args);
  if (args->thread == 0) {
      for (int i=1; i<args->nThreads; i++) { 
          maxDelta += args->deltaThreads[i];
      }
#ifdef MPI_SUPPORT
      MPI_Allreduce(MPI_IN_PLACE, &maxDelta, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
#endif
  }
  Barrier(args);

  if (datacheck) { 
     PRINT("  %7.3f  %5.2f  %5.2f  %7.0le", deltaSec * 1.0E3, algBw, busBw,
         maxDelta);
  } else {
     PRINT("  %7.3f  %5.2f  %5.2f  \tN/A", deltaSec * 1.0E3, algBw, busBw);
  }

  args->bw[0] += busBw;
  args->bw_count[0]++;
}

void TimeTest(struct threadArgs_t* args, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName, int root, int inPlace) {
  size_t size;
  int nranks = args->nProcs*args->nGpus*args->nThreads; 
  size_t count, sendCount, recvCount, sendInplaceOffset, recvInplaceOffset, procSharedCount;
  int sameExpected;
  for (size = args->minbytes; size<=args->maxbytes; size = ((args->stepfactor > 1) ? size*args->stepfactor : size+args->stepbytes)) { 
      count = size / wordSize(type);
      getCollByteCount(&sendCount, &recvCount, &sendInplaceOffset, &recvInplaceOffset, &procSharedCount, &sameExpected, (size_t)count, (size_t)nranks);

      args->nbytes = count * wordSize(type); 
      args->sendBytes = sendCount * wordSize(type); 
      args->expectedBytes = recvCount * wordSize(type);
      args->sendInplaceOffset = sendInplaceOffset * wordSize(type);
      args->recvInplaceOffset = recvInplaceOffset * wordSize(type);

      print_line_header(args->nbytes, count, typeName, opName, root);

      BenchTime(args, type, op, root, 0);
      if (inPlace) BenchTime(args, type, op, root, 1);
      PRINT("\n");
  }
}


void* threadRunTests(void* args) {
  RunTest((struct threadArgs_t*)args, ncclroot, (ncclDataType_t)nccltype, nccltypestring, (ncclRedOp_t)ncclop, ncclopstring);

  return NULL;
}

void AllocateBuffs(void **sendbuff, size_t sendBytes, void **recvbuff, size_t recvBytes, void **expected, void **expectedHost, size_t nbytes, int nranks, int sameExpected) {
    static int is_first = 1;
    static void *cached_ptr = NULL;
    static void *cached_hostptr = NULL;

    CUDACHECK(hipMalloc(sendbuff, sendBytes));
    //work around for inline reduce scatter where recv count is smaller that send count
    CUDACHECK(hipMalloc(recvbuff, (sendBytes > recvBytes) ? sendBytes : recvBytes));

    if (is_first || !sameExpected) {
        *expectedHost = malloc(recvBytes);
        CUDACHECK(hipHostRegister(*expectedHost, recvBytes, hipHostRegisterPortable | hipHostRegisterMapped));
        CUDACHECK(hipHostGetDevicePointer(expected, *expectedHost, 0));
        cached_ptr = *expected;
        cached_hostptr = *expectedHost;
        is_first = 0;
    } else {
        *expected = cached_ptr;
        *expectedHost = cached_hostptr;
    }
}
 
int ncclstringtotype (char *str) { 
    int type;

    printf("input %s", str);

    if (!strcmp(str, "int8")) type = (int)ncclInt8; 
    else if (!strcmp(str, "uint8")) type = (int)ncclUint8; 
    else if (!strcmp(str, "int32")) type = (int)ncclInt32; 
    else if (!strcmp(str, "uint32")) type = (int)ncclUint32;
     else if (!strcmp(str, "int64")) type = (int)ncclInt64; 
    else if (!strcmp(str, "uint64")) type = (int)ncclUint64; 
    else if (!strcmp(str, "half")) type = (int)ncclHalf; 
    else if (!strcmp(str, "float")) type = (int)ncclFloat;
    else if (!strcmp(str, "double")) type = (int)ncclDouble;
    else if (!strcmp(str, "all")) type = -1;
    else printf("invalid type, defaulting to uint32... \n"); 

    if(type != (int)ncclUint32) { 
      strcpy(nccltypestring, str);
    }

    return type;
}

int ncclstringtoop (char *str) { 
    int op;

    printf("input %s", str);

    if (!strcmp(str, "sum")) op = (int)ncclSum; 
    else if (!strcmp(str, "prod")) op = (int)ncclProd; 
    else if (!strcmp(str, "min")) op = (int)ncclMin; 
    else if (!strcmp(str, "max")) op = (int)ncclMax; 
    else if (!strcmp(str, "all")) op = -1;
    else printf("invalid op, defaulting to sum... \n"); 

    if(op != (int)ncclSum) { 
      strcpy(ncclopstring, str);
    }

    return op;
}

int main(int argc, char* argv[]) {
 int nThreads = 1, nGpus = 1;
 size_t minBytes = 32*1024*1024, maxBytes = 32*1024*1024, stepBytes = 1*1024*1024, stepFactor = 1;
 int longindex;
 int nProcs = 1, proc = 0;
 int localRank = 0;
 char hostname[1024];
 getHostName(hostname, 1024);
 
 static struct option longopts[] = {
    {"nthreads", required_argument, 0, 't'}, 
    {"ngpus", required_argument, 0, 'g'}, 
    {"minbytes", required_argument, 0, 'b'}, 
    {"maxbytes", required_argument, 0, 'e'}, 
    {"stepbytes", required_argument, 0, 'i'},
    {"stepfactor", required_argument, 0, 'f'},
    {"iters", required_argument, 0, 'n'},
    {"check", required_argument, 0, 'c'},
    {"op", required_argument, 0, 'o'},
    {"datatype", required_argument, 0, 'd'},
    {"root", required_argument, 0, 'r'},
    {"help", no_argument, 0, 'h'}
 };

 while(1) {
      int c;
      c = getopt_long(argc, argv, "t:g:b:e:i:f:n:c:o:d:r:h", longopts, &longindex);

      if (c == -1)
         break;

      switch(c) {
         case 't':
             nThreads = strtol(optarg, NULL, 0);
             break;
         case 'g':
             nGpus = strtol(optarg, NULL, 0);
             break;
         case 'b':
             minBytes = (size_t)parsesize(optarg);
             break;
         case 'e':
             maxBytes = (size_t)parsesize(optarg);
             break;
         case 'i':
             stepBytes = strtol(optarg, NULL, 0);
             break;
         case 'f':
             stepFactor = strtol(optarg, NULL, 0);
             break;
	     case 'n':
	         iters = (int)strtol(optarg, NULL, 0);
	         break;
	     case 'c':
	         datacheck = (int)strtol(optarg, NULL, 0);
	         break;
	     case 'o':
	         ncclop = ncclstringtoop(optarg);
	         break;
	     case 'd':
	         nccltype = ncclstringtotype(optarg);
	         break;
	     case 'r':
	         ncclroot = strtol(optarg, NULL, 0);
	         break;
         case 'h':
	         printf("USAGE: ./test [-t,--nthreads <num threads>] [-g,--ngpus <gpus per thread>] [-b,--minbytes <min size in bytes>] [-e,--maxbytes <max size in bytes>] [-i,--stepbytes <increment size>]"
	         " [-f,--stepfactor <increment factor>] [-n,--iters <iteration count>] [-c,--check <0/1>] [-o,--op <sum/prod/min/max/all>] [-d,--datatype <nccltype/all>] [-r,--root <root>] [-h,--help]\n");
	         return 0;
	     default: 
	         printf("invalid option \n");
	         printf("USAGE: ./test [-t,--nthreads <num threads>] [-g,--ngpus <gpus per thread>] [-b,--minbytes <min size in bytes>] [-e,--maxbytes <max size in bytes>] [-i,--stepbytes <increment size>]"
	         " [-f,--stepfactor <increment factor>] [-n,--iters <iteration count>] [-c, --check <0/1>] [-o,--op <sum/prod/min/max/all>] [-d,--datatype <nccltype/all>] [-r,--root <root>] [-h,--help]\n");
	         return 0;
      }
  }


#ifdef MPI_SUPPORT
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &nProcs);
  MPI_Comm_rank(MPI_COMM_WORLD, &proc);
  uint64_t hostHashs[nProcs];
  hostHashs[proc] = getHostHash(hostname);
  MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD);
  for (int p=0; p<nProcs; p++) {
    if (p == proc) break;
    if (hostHashs[p] == hostHashs[proc]) localRank++;
  }
#endif
#ifdef MPI_TRANSPORT
  ncclMpiHook(MPI_COMM_WORLD);
#endif
  is_main_thread = (proc == 0) ? 1 : 0;

  if (proc == 0) { 
      printf("nThread %d nGpus %d minBytes %ld maxBytes %ld step: %ld(%s) iters: %d validation: %d \n", nThreads, nGpus, minBytes, maxBytes, 
      			(stepFactor > 1)?stepFactor:stepBytes, (stepFactor > 1)?"factor":"bytes", iters, datacheck);
  }

  ncclUniqueId ncclId;
  if (proc == 0) {
    NCCLCHECK(ncclGetUniqueId(&ncclId));
  }
#ifdef MPI_SUPPORT
  MPI_Bcast(&ncclId, sizeof(ncclId), MPI_BYTE, 0, MPI_COMM_WORLD);
#endif
  hipStream_t streams[nGpus*nThreads];
  void* sendbuffs[nGpus*nThreads];
  void* recvbuffs[nGpus*nThreads];
  void* expected[nGpus*nThreads];
  void* expectedHost[nGpus*nThreads];
  void *procSharedHost, *procShared;
  size_t sendBytes, recvBytes, procSharedBytes, sendInplaceOffset, recvInplaceOffset; 
  int sameExpected;

  getCollByteCount(&sendBytes, &recvBytes, &sendInplaceOffset, &recvInplaceOffset, &procSharedBytes, &sameExpected, (size_t)maxBytes, (size_t)nProcs*nGpus*nThreads);

  NCCLCHECK(ncclGroupStart());
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nThreads*nGpus);
  for (int i=0; i<nGpus*nThreads; i++) {
    CUDACHECK(hipSetDevice(localRank*nThreads*nGpus+i));
    AllocateBuffs(sendbuffs+i, sendBytes, recvbuffs+i, recvBytes, expected+i, expectedHost+i, (size_t)maxBytes, nProcs*nThreads*nGpus, sameExpected);
    CUDACHECK(hipStreamCreate(streams+i));
    NCCLCHECK(ncclCommInitRank(comms+i, nProcs*nThreads*nGpus, ncclId, proc*nThreads*nGpus+i));
  }
  NCCLCHECK(ncclGroupEnd());

  if (procSharedBytes > 0) { 
      procSharedHost = malloc(procSharedBytes);
      CUDACHECK(hipHostRegister(procSharedHost, procSharedBytes, hipHostRegisterPortable | hipHostRegisterMapped));
      CUDACHECK(hipHostGetDevicePointer(&procShared, procSharedHost, 0));
  }

  PRINT("# Using devices\n");
  for (int p=0; p<nProcs; p++) {
    if (p == proc) {
      for (int i=0; i<nThreads*nGpus; i++) {
        int cudaDev;
        int rank;
        hipDeviceProp_t prop;
        NCCLCHECK(ncclCommCuDevice(comms[i], &cudaDev));
        NCCLCHECK(ncclCommUserRank(comms[i], &rank));
        CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
        printf("#   Rank %2d on %10s device %2d [0x%02x] %s\n", rank, hostname, cudaDev,
            prop.pciBusID, prop.name);
        fflush(stdout);
      }
    }
#ifdef MPI_SUPPORT
    MPI_Barrier(MPI_COMM_WORLD);
#endif
    printf("");
    fflush(stdout);
  }

  int errors[nThreads];
  double bw[nThreads];
  double delta[nThreads];
  int bw_count[nThreads];
  for (int t=0; t<nThreads; t++) {
    bw[t] = 0.0;
    errors[t] = bw_count[t] = 0;
  }

  PRINT("\n");
  print_header();

  int* sync = (int*)malloc(sizeof(int)*2);
  sync[0] = 0;
  sync[1] = 0;

  pthread_t threads[nThreads-1];
  struct threadArgs_t args[nThreads];
  for (int t=nThreads-1; t>=0; t--) {
    args[t].minbytes=minBytes;
    args[t].maxbytes=maxBytes;
    args[t].stepbytes=stepBytes;
    args[t].stepfactor=stepFactor;

    args[t].nProcs=nProcs;
    args[t].proc=proc;
    args[t].nThreads=nThreads;
    args[t].thread=t;
    args[t].nGpus=nGpus;
    args[t].sendbuffs = sendbuffs+t*nGpus;
    args[t].recvbuffs = recvbuffs+t*nGpus;
    args[t].comms=comms+t*nGpus;
    args[t].streams=streams+t*nGpus;

    args[t].expectedHost = expectedHost + t*nGpus;
    args[t].expected = expected + t*nGpus;
    args[t].procSharedHost = procSharedHost; 
    args[t].procShared = procShared; 
    args[t].sync = (volatile int*)sync;
    args[t].deltaThreads = delta;
    args[t].deltaHost = (delta + t);
    CUDACHECK(hipHostRegister(args[t].deltaHost, sizeof(double), 0));
    CUDACHECK(hipHostGetDevicePointer(&args[t].delta, args[t].deltaHost, 0));
    args[t].errors=errors+t;
    args[t].bw=bw+t;
    args[t].bw_count=bw_count+t;
    if (t)
      pthread_create(threads+t-1, NULL, threadRunTests, args+t);
    else { 
      RunTest((struct threadArgs_t*)args, ncclroot, (ncclDataType_t)nccltype, nccltypestring, (ncclRedOp_t)ncclop, ncclopstring);
    }
  }
  // Wait for other threads
  for (int t=1; t<nThreads; t++) {
    pthread_join(threads[t-1], NULL);
    errors[0] += errors[t];
    bw[0] += bw[t];
    bw_count[0] += bw_count[t];
  }

#ifdef MPI_SUPPORT
    MPI_Allreduce(MPI_IN_PLACE, &errors[0], 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
#endif

  for(int i=0; i<nGpus*nThreads; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  char* str = getenv("NCCL_TESTS_MIN_BW");
  double check_avg_bw = str ? atof(str) : -1;
  bw[0] /= bw_count[0];

  PRINT(" Out of bounds values : %d %s\n", errors[0], errors[0] ? "FAILED" : "OK");
  PRINT(" Avg bus bandwidth    : %g %s\n", bw[0], check_avg_bw == -1 ? "" : (bw[0] < check_avg_bw ? "FAILED" : "OK"));
  PRINT("\n");
#ifdef MPI_SUPPORT
  MPI_Finalize();
#endif
  if (errors[0] || bw[0] < check_avg_bw)
    exit(EXIT_FAILURE);
  else 
    exit(EXIT_SUCCESS);
}
