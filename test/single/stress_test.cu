#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <sys/time.h>

#include "nccl.h"
#include "test_utilities.h"

#define MAX_POW2 27 // 128 MB
#define NREPS 4
//#define CHECK 1

int nDev;
int* devList;
hipStream_t* streams;
char** sendbuff;
char** recvbuff;
char* reference;
char* results;

double CheckTypeDelta(int type, char* devmem, char* ref, int count) {
  switch(type) {
    case ncclInt8:   return CheckDelta<int8_t>((int8_t*)devmem, (int8_t*)ref, count);
    case ncclUint8:  return CheckDelta<uint8_t>((uint8_t*)devmem, (uint8_t*)ref, count);
    case ncclInt32:  return CheckDelta<int32_t>((int32_t*)devmem, (int32_t*)ref, count);
    case ncclUint32: return CheckDelta<uint32_t>((uint32_t*)devmem, (uint32_t*)ref, count);
    case ncclHalf:   return CheckDelta<half>((half*)devmem, (half*)ref, count);
    case ncclFloat:  return CheckDelta<float>((float*)devmem, (float*)ref, count);
    case ncclDouble: return CheckDelta<double>((double*)devmem, (double*)ref, count);
    case ncclInt64:  return CheckDelta<int64_t>((int64_t*)devmem, (int64_t*)ref, count);
    case ncclUint64: return CheckDelta<uint64_t>((uint64_t*)devmem, (uint64_t*)ref, count);
  }
  return 0.0;
}

void AccumulateType(int type, char* ref, char*devmem, int count, ncclRedOp_t op) {
  switch(type) {
    case ncclInt8:   return Accumulate<int8_t>((int8_t*)ref, (int8_t*)devmem, count, op);
    case ncclUint8:  return Accumulate<uint8_t>((uint8_t*)ref, (uint8_t*)devmem, count, op);
    case ncclInt32:  return Accumulate<int32_t>((int32_t*)ref, (int32_t*)devmem, count, op);
    case ncclUint32: return Accumulate<uint32_t>((uint32_t*)ref, (uint32_t*)devmem, count, op);
    case ncclHalf:   return Accumulate<half>((half*)ref, (half*)devmem, count, op);
    case ncclFloat:  return Accumulate<float>((float*)ref, (float*)devmem, count, op);
    case ncclDouble: return Accumulate<double>((double*)ref, (double*)devmem, count, op);
    case ncclInt64:  return Accumulate<int64_t>((int64_t*)ref, (int64_t*)devmem, count, op);
    case ncclUint64: return Accumulate<uint64_t>((uint64_t*)ref, (uint64_t*)devmem, count, op);
  }
}

void RandomizeType(int type, char* devmem, int count, int seed) {
  switch(type) {
    case ncclInt8:   return Randomize((int8_t*)devmem, count, seed);
    case ncclUint8:  return Randomize((uint8_t*)devmem, count, seed);
    case ncclInt32:  return Randomize((int32_t*)devmem, count, seed);
    case ncclUint32: return Randomize((uint32_t*)devmem, count, seed);
    case ncclHalf:   return Randomize((half*)devmem, count, seed);
    case ncclFloat:  return Randomize((float*)devmem, count, seed);
    case ncclDouble: return Randomize((double*)devmem, count, seed);
    case ncclInt64:  return Randomize((int64_t*)devmem, count, seed);
    case ncclUint64: return Randomize((uint64_t*)devmem, count, seed);
  }
}

typedef int (*test_func_t)(int, ncclDataType_t, int, int, int, ncclComm_t*);

int testBcast(int count, ncclDataType_t type, int op, int root, int nranks, ncclComm_t *comms) {
  int errors = 0;
  size_t nbytes = count*wordSize(type);
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    if (i == root) {
      RandomizeType(type, sendbuff[i], count, i);
      CUDACHECK(hipMemcpy(reference, sendbuff[i], nbytes, hipMemcpyDeviceToHost));
    } else {
      CUDACHECK(hipMemset(recvbuff[i], 0, nbytes));
    }
  }
#endif

  for (int rep=0; rep<NREPS; ++rep) {
    for (int i=0; i<nranks; ++i) {
      CUDACHECK(hipSetDevice(devList[i]));
      ncclBcast((i == root) ? sendbuff[i] : recvbuff[i], count, (ncclDataType_t)type, root, comms[i], streams[i]);
    }
  }
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamSynchronize(streams[i]));
    double delta = CheckDelta<char>((i==root) ? sendbuff[i] : recvbuff[i], reference, nbytes);
    if (delta) errors++;
    if (delta) printf("Bcast size %d, type %d, root %d : delta %g\n", count, type, root, delta);
  }
#endif
  return errors;
}

int testAllGather(int count, ncclDataType_t type, int op, int root, int nranks, ncclComm_t *comms) {
  int errors = 0;
  int sendcount = (count + nranks - 1) / nranks;
  int recvcount = sendcount * nranks;
  int sendnbytes = sendcount*wordSize(type);
  int recvnbytes = recvcount*wordSize(type);
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    RandomizeType(type, sendbuff[i], sendcount, i);
    CUDACHECK(hipMemcpy(reference+sendnbytes*i, sendbuff[i], sendnbytes, hipMemcpyDeviceToHost));
    CUDACHECK(hipMemset(recvbuff[i], 0, recvnbytes));
  }
#endif

  for (int rep=0; rep<NREPS; ++rep) {
    for (int i=0; i<nranks; ++i) {
      CUDACHECK(hipSetDevice(devList[i]));
      ncclAllGather(sendbuff[i], recvbuff[i], sendcount, (ncclDataType_t)type, comms[i], streams[i]);
    }
  }
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamSynchronize(streams[i]));
    double delta = CheckTypeDelta(type, recvbuff[i], reference, recvcount);
    if (delta) {
      errors++;
      CUDACHECK(hipMemcpy(results, recvbuff[i], recvnbytes, hipMemcpyDeviceToHost));
      printf("Allgather size %d, type %d : delta %g, new %g\n", sendcount, type, delta);
      for (int c=1; c<count; c++) {
	if (type == ncclFloat) {
          float res = *((float*)results+c), ref = *((float*)reference+c);
          if (fabs(res-ref) > deltaMaxValue(type, 0)*nranks) printf("[%d/%3d] %f != %f (+%f)\n", i, c, res, ref, (ref-res)/ref);
        } else if (type == ncclDouble) {
          double res = *((double*)results+c), ref = *((double*)reference+c);
          if (fabs(ref-res) > deltaMaxValue(type, 0)*nranks) printf("[%d/%3d] %g != %g (+%g)\n", i, c, res, ref, (ref-res)/ref);
        } else if (c*8 < count*wordSize(type)) {
          uint64_t res = *((uint64_t*)results+c), ref = *((uint64_t*)reference+c);
          if (res != ref) printf("[%d/%3d] %16lX != %16lX\n", i, c, res, ref);
        }
      }
    }
  }
#endif
  return errors;
}

int testAllReduce(int count, ncclDataType_t type, int op, int root, int nranks, ncclComm_t *comms) {
  int errors = 0;
  int nbytes = count*wordSize(type);
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    RandomizeType(type, sendbuff[i], count, i);
    if(i == 0) {
      CUDACHECK(hipMemcpy(reference, sendbuff[i], nbytes, hipMemcpyDeviceToHost));
    } else {
      AccumulateType(type, reference, sendbuff[i], count, (ncclRedOp_t)op);
    }
    CUDACHECK(hipMemset(recvbuff[i], 0, nbytes));
  }
#endif

  for (int rep=0; rep<NREPS; ++rep) {
    for (int i=0; i<nranks; ++i) {
      CUDACHECK(hipSetDevice(devList[i]));
      ncclAllReduce(sendbuff[i], recvbuff[i], count, (ncclDataType_t)type, (ncclRedOp_t)op, comms[i], streams[i]);
    }
  }
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamSynchronize(streams[i]));
    double delta = CheckTypeDelta(type, recvbuff[i], reference, count);
    if (delta > deltaMaxValue(type, 1)*nranks) {
      errors++;
      CUDACHECK(hipMemcpy(results, recvbuff[i], nbytes, hipMemcpyDeviceToHost));
      printf("Allreduce size %d, type %d, op %d : delta %g\n", count, type, op, delta);
#ifdef DEBUG_DETAILS
      for (int c=1; c<count; c++) {
	if (type == ncclFloat) {
          float res = *((float*)results+c), ref = *((float*)reference+c);
          if (fabs(res-ref) > deltaMaxValue(type, 1)*nranks) printf("[%d/%3d] %f != %f (+%f)\n", i, c, res, ref, (ref-res)/ref);
        } else if (type == ncclDouble) {
          double res = *((double*)results+c), ref = *((double*)reference+c);
          if (fabs(ref-res) > deltaMaxValue(type, 1)*nranks) printf("[%d/%3d] %g != %g (+%g)\n", i, c, res, ref, (ref-res)/ref);
        } else if (c*8 < count*wordSize(type)) {
          uint64_t res = *((uint64_t*)results+c), ref = *((uint64_t*)reference+c);
          if (res != ref) printf("[%d/%3d] %16lX != %16lX\n", i, c, res, ref);
        }
      }
#endif
    }
  }
#endif
  return errors;
}

int testReduce(int count, ncclDataType_t type, int op, int root, int nranks, ncclComm_t *comms) {
  int errors = 0;
  int nbytes = count*wordSize(type);
#ifdef CHECK
  for (int i=0; i<nranks; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    RandomizeType(type, sendbuff[i], count, i);
    if(i == 0) {
      CUDACHECK(hipMemcpy(reference, sendbuff[i], nbytes, hipMemcpyDeviceToHost));
    } else {
      AccumulateType(type, reference, sendbuff[i], count, (ncclRedOp_t)op);
    }
    if (i == root) {
      CUDACHECK(hipMemset(recvbuff[i], 0, nbytes));
    }
  }
#endif

  for (int rep=0; rep<NREPS; ++rep) {
    for (int i=0; i<nranks; ++i) {
      CUDACHECK(hipSetDevice(devList[i]));
      ncclReduce(sendbuff[i], recvbuff[i], count, (ncclDataType_t)type, (ncclRedOp_t)op, root, comms[i], streams[i]);
    }
  }
#ifdef CHECK
  for (int i=0; i<nranks; i++) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamSynchronize(streams[i]));
    if (i == root) {
      double delta = CheckTypeDelta(type, recvbuff[i], reference, count);
      if (delta > deltaMaxValue(type, 1)*nranks) {
        errors++;
        CUDACHECK(hipMemcpy(results, recvbuff[i], nbytes, hipMemcpyDeviceToHost));
        printf("Reduce size %d, type %d, op %d, root %d : delta %g\n", count, type, op, root, delta);
      }
    }
  }
#endif
  return errors;
}

#define NCCL_PRIMS 5

test_func_t ncclPrims[NCCL_PRIMS] = {
  testBcast,
  testReduce,
  testAllReduce,
  testAllGather,
  NULL,//testReduceScatter
};

int ncclTest(ncclComm_t ** comms) {
  int errors = 0;
  int nccl_prim = rand() % NCCL_PRIMS;
  // Use MAX_POW2-3 because datatypes are up to 8-bytes wide
  int size_pow2 = rand() % (MAX_POW2-3); 
  int size = (1<<size_pow2) + rand() % (1<<size_pow2);
  ncclDataType_t type = (ncclDataType_t)( rand() % ncclNumTypes );
  int op = rand() % ncclNumOps;
  int commidx = rand() % nDev;
  int nranks = commidx + 1;
  int root = rand() % nranks;
  if (type == 2) return 0; // ncclHalf not supported
  if (ncclPrims[nccl_prim]) {
    printf("Prim %d size %d type %d op %d nranks %d root %d\n", nccl_prim, size, type, op, commidx+1, root);
    errors += ncclPrims[nccl_prim](size, type, op, root, nranks, comms[commidx]);
  }
  return errors;
}

void usage() {
  printf("Tests all nccl primitives.\n"
      "    Usage: stress_test [time in sec] [number of GPUs]"
      "[GPU 0] [GPU 1] ...\n\n");
}

int main(int argc, char* argv[]) {
  int nVis = 0;
  CUDACHECK(hipGetDeviceCount(&nVis));

  int T = 0;
  if (argc > 1) {
    int t = sscanf(argv[1], "%d", &T);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  } else {
    printf("Error: must specify at least time in seconds!\n\n");
    usage();
    exit(EXIT_FAILURE);
  }

  nDev = nVis;
  if (argc > 2) {
    int t = sscanf(argv[2], "%d", &nDev);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  }
  devList = (int*)malloc(sizeof(int)*nDev);
  for (int i = 0; i < nDev; ++i)
    devList[i] = i % nVis;

  if (argc > 3) {
    if (argc - 3 != nDev) {
      printf("Error: insufficient number of GPUs in list\n\n");
      usage();
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < nDev; ++i) {
      int t = sscanf(argv[3 + i], "%d", devList + i);
      if (t == 0) {
        printf("Error: %s is not an integer!\n\n", argv[2 + i]);
        usage();
        exit(EXIT_FAILURE);
      }
    }
  }

  ncclComm_t** comms = (ncclComm_t**)malloc(sizeof(ncclComm_t*)*nDev);
  for (int i=0; i<nDev; i++) {
    comms[i] = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
    NCCLCHECK(ncclCommInitAll(comms[i], i+1, devList));
  }

  streams = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  sendbuff = (char**)malloc(sizeof(char*)*nDev);
  recvbuff = (char**)malloc(sizeof(char*)*nDev);
  for(int i=0; i<nDev; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamCreate(&streams[i]));
    CUDACHECK(hipMalloc(sendbuff+i, 1<<MAX_POW2));
    CUDACHECK(hipMalloc(recvbuff+i, 1<<MAX_POW2));
  }

  reference = (char*)malloc(1<<MAX_POW2);
  results = (char*)malloc(1<<MAX_POW2);
 
  struct timeval tv;
  gettimeofday(&tv, NULL);
  int sec_now = tv.tv_sec;
  int sec_start = sec_now;
  srand(sec_start);
  int testcount = 0;
  int errors = 0;

  printf("==== Test starting ====\n");
  while (sec_now <= sec_start + T) {
    errors += ncclTest(comms);
    gettimeofday(&tv, NULL);
    sec_now = tv.tv_sec;
    testcount++;
  }

#ifndef CHECK
  for(int i=0; i<nDev; ++i) {
    CUDACHECK(hipSetDevice(devList[i]));
    CUDACHECK(hipStreamSynchronize(streams[i]));
  }
#endif

  printf("==== Test done ====\n");
  printf("%d tests done\n", testcount);
  printf("%d errors\n", errors);
  for (int i=0; i<nDev; i++) {
    for(int j=0; j<i; j++) {
      ncclCommDestroy(comms[i][j]);
    }
    free(comms[i]);
  }
  free(comms);

  exit(errors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

