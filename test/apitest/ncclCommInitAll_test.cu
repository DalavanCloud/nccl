class ncclCommInitAll_test : public ::testing::Test {
  protected:
    ncclComm_t* comms = NULL;
    int* devList = NULL;
    int nVis = 0;
    virtual void SetUp() {
        ASSERT_EQ(hipSuccess, hipGetDeviceCount(&nVis));
        ASSERT_GT(nVis, 1);
        comms = (ncclComm_t*)calloc(nVis, sizeof(ncclComm_t));
        devList = (int*)calloc(nVis, sizeof(int));
        for (int i = 0; i < nVis; i++) {
            devList[i] = i;
        }
    };
    virtual void TearDown() {
        if (NULL != devList) {
            free(devList);
            devList = NULL;
        }
        if (NULL != comms) {
            for (int i = 0; i < nVis; ++i) {
                ncclCommDestroy(comms[i]);
                comms[i] = NULL;
            }
            free(comms);
            comms = NULL;
        }
    };
};
TEST_F(ncclCommInitAll_test, basic) {
    ASSERT_EQ(ncclSuccess, ncclCommInitAll(comms, nVis, devList));
};
// 1.
TEST_F(ncclCommInitAll_test, comms_null) {
    ASSERT_EQ(ncclInvalidArgument, ncclCommInitAll(NULL, nVis, devList));
};
// 2.
TEST_F(ncclCommInitAll_test, ndev_0) {
    ASSERT_EQ(ncclUnsupportedDeviceCount, ncclCommInitAll(comms, 0, devList));
};
TEST_F(ncclCommInitAll_test, ndev_negative) {
    ASSERT_EQ(ncclUnsupportedDeviceCount, ncclCommInitAll(comms, -1, devList));
};
// TODO: fix this
TEST_F(ncclCommInitAll_test, DISABLED_ndev_toomany) {
    int ndev = 100;
    ncclComm_t* pcomms = (ncclComm_t*)calloc(ndev, sizeof(ncclComm_t));
    EXPECT_EQ(ncclUnsupportedDeviceCount, ncclCommInitAll(pcomms, ndev, NULL));
    if (NULL != pcomms) {
        for (int i = 0; i < ndev; ++i) {
            ncclCommDestroy(pcomms[i]);
            pcomms[i] = NULL;
        }
        free(pcomms);
        pcomms = NULL;
    }
};
#if 0 // useless?
TEST_F(ncclCommInitAll_test, ndev_toomany_and_devList_allZero) {
    int ndev = 100;
    comms = (ncclComm_t*)calloc(ndev, sizeof(ncclComm_t));
    devList = (int*)calloc(ndev, sizeof(int));
    ASSERT_EQ(ncclUnsupportedDeviceCount,
              ncclCommInitAll(comms, ndev, devList));
};
#endif
// 3.
TEST_F(ncclCommInitAll_test, devList_null) {
    ASSERT_EQ(ncclSuccess, ncclCommInitAll(comms, nVis, NULL));
};
TEST_F(ncclCommInitAll_test, devList_nonexist) {
    int* badDevList = (int*)calloc(nVis, sizeof(int));
    for (int i = 0; i < nVis; ++i) {
        badDevList[i] = 1000 + i;
    }
    ASSERT_EQ(ncclInvalidDeviceIndex, ncclCommInitAll(comms, nVis, badDevList));
    free(badDevList);
};
// EOF
