#include "hip/hip_runtime.h"
#include "ncclCommon_test.cuh"
template <typename DT>
class ncclAllGather_test : public ncclCommon_test<DT> {};
TYPED_TEST_CASE(ncclAllGather_test, testDataTypes);
// typical usage.
TYPED_TEST(ncclAllGather_test, basic) {
    for (int i = 0; i < this->nVis; ++i) {
        ASSERT_EQ(hipSuccess, hipSetDevice(i)) << "i" << i << ", "
                                                 << std::endl;
        ASSERT_EQ(ncclSuccess,
                  ncclAllGather(this->sendbuffs[i], this->recvbuffs[i],
                                std::min(this->N/this->nVis, 1024 * 1024),
                                this->DataType(), this->comms[i], this->streams[i]))
            << "i" << i << ", " << std::endl;
    }
};
TYPED_TEST(ncclAllGather_test, host_mem) {
    for (int i = 0; i < this->nVis; ++i) {
        ASSERT_EQ(hipSuccess, hipSetDevice(i)) << "i" << i << ", "
                                                 << std::endl;
        EXPECT_EQ(ncclInvalidDevicePointer,
                  ncclAllGather(this->sendbuffs_host[i], this->recvbuffs_host[i],
                                std::min(this->N/this->nVis, 1024 * 1024),
                                this->DataType(), this->comms[i], this->streams[i]))
            << "i" << i << ", " << std::endl;
    }
};
TYPED_TEST(ncclAllGather_test, DISABLED_pinned_mem) {
    for (int i = 0; i < this->nVis; ++i) {
        ASSERT_EQ(hipSuccess, hipSetDevice(i)) << "i" << i << ", "
                                                 << std::endl;
        EXPECT_EQ(ncclSuccess,
                  ncclAllGather(this->sendbuffs_pinned[i], this->recvbuffs_pinned[i],
                                std::min(this->N/this->nVis, 1024 * 1024),
                                this->DataType(), this->comms[i], this->streams[i]))
            << "i" << i << ", " << std::endl;
    }
};
// sendbuff
TYPED_TEST(ncclAllGather_test, sendbuf_null) {
    int i = 0;
    EXPECT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(NULL, this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), this->comms[i], this->streams[i]));
};
TYPED_TEST(ncclAllGather_test, sendbuf_wrong) {
    int i = 0, j = 1;
    ASSERT_EQ(hipSuccess, hipSetDevice(i));
    EXPECT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(this->sendbuffs[j], this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(),
                            this->comms[i], this->streams[i]));
};
// recvbuff
TYPED_TEST(ncclAllGather_test, recvbuf_null) {
    int i = 0;
    ASSERT_EQ(hipSuccess, hipSetDevice(i));
    EXPECT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(this->sendbuffs[i], NULL,
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), this->comms[i], this->streams[i]));
}
// sendbuff and recvbuff not on the same device
TYPED_TEST(ncclAllGather_test, sendbuff_recvbuff_diff_device) {
    int i = 0, j = 1;
    ASSERT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(this->sendbuffs[i], this->recvbuffs[j],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), this->comms[i], this->streams[i]));
};
// N
TYPED_TEST(ncclAllGather_test, DISABLED_N_zero) {
    for (int i = 0; i < this->nVis; ++i) {
        ASSERT_EQ(hipSuccess, hipSetDevice(i)) << "i" << i << ", "
                                                 << std::endl;
        ASSERT_EQ(ncclSuccess,
                  ncclAllGather(this->sendbuffs[i], this->recvbuffs[i], 0,
                                this->DataType(), this->comms[i], this->streams[i]))
            << "i" << i << ", " << std::endl;
    }
};
// data type
TYPED_TEST(ncclAllGather_test, DataType_wrong) {
    int i = 0;
    ASSERT_EQ(ncclInvalidType,
              ncclAllGather(this->sendbuffs[i], this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            nccl_NUM_TYPES, this->comms[i], this->streams[i]));
};
// comm
TYPED_TEST(ncclAllGather_test, comm_null) {
    int i = 0;
    ASSERT_EQ(ncclInvalidArgument,
              ncclAllGather(this->sendbuffs[i], this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), NULL, this->streams[i]));
};
TYPED_TEST(ncclAllGather_test, comm_wrong) {
    int i = 0, j = 1;
    ASSERT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(this->sendbuffs[i], this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), this->comms[j], this->streams[i]));
};
// STREAM can be NULL.
// stream on a diff device
TYPED_TEST(ncclAllGather_test, DISABLED_stream_wrong) {
    int i = 0, j = 1;
    ASSERT_EQ(ncclInvalidDevicePointer,
              ncclAllGather(this->sendbuffs[i], this->recvbuffs[i],
                            std::min(this->N/this->nVis, 1024 * 1024),
                            this->DataType(), this->comms[i], this->streams[j]));
};
// EOF
