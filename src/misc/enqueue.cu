/*************************************************************************
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "enqueue.h"
#include "common_coll.h"

ncclResult_t ncclLaunchCooperativeKernelMultiDevice(struct hipLaunchParams *paramsList, int* cudaDevs, int numDevices, int cgMode) {
#if __CUDACC_VER_MAJOR__ >= 9
  if (cgMode & 0x01) {
    CUDACHECK(hipLaunchCooperativeKernelMultiDevice(paramsList, numDevices,
          // These flags are to reduce the latency of using this API
          hipCooperativeLaunchMultiDeviceNoPreSync|hipCooperativeLaunchMultiDeviceNoPostSync));
    return ncclSuccess;
  }
#endif
  int savedDev;
  CUDACHECK(hipGetDevice(&savedDev));
  for (int i = 0; i < numDevices; i++) {
    struct hipLaunchParams* params = paramsList+i;
    CUDACHECK(hipSetDevice(cudaDevs[i]));
    CUDACHECK(hipLaunchKernel(reinterpret_cast<const void*>(params->func), params->gridDim, params->blockDim, params->args, params->sharedMem, params->stream));
  }
  CUDACHECK(hipSetDevice(savedDev));
  return ncclSuccess;
}

ncclResult_t ncclCpuBarrierCheckin(ncclComm_t comm) {
  if (comm->nRanks == 1) return ncclSuccess;
  if (comm->launchMode == ncclComm::GROUP) {
    // Enqueue stream dependency
    CUDACHECK(hipEventRecord(comm->doneEvent, comm->userStream));
    CUDACHECK(hipStreamWaitEvent(comm->ncclStream, comm->doneEvent, 0));
  } else {
    if (comm->userStream != comm->ncclStream) {
      CUDACHECK(hipStreamWaitEvent(comm->userStream, comm->doneEvent, 0));
    }
  }
  // Notify I'm ready
  volatile int* ptr = (volatile int*)(comm->intraBarrier+comm->intraPhase);
  int val = *ptr;
  bool done = false;
  while (done == false) {
    if (val >= comm->intraRanks) {
      WARN("Trying to launch too many collectives");
      return ncclInvalidUsage;
    }
    if (val+1 == comm->intraRanks) {
      if (comm->launchMode == ncclComm::GROUP) {
        // I'm the last. Launch all operations.
        ncclLaunchCooperativeKernelMultiDevice(comm->intraParams, comm->intraCudaDevs, comm->intraRanks, *comm->intraCGMode);
      }
      // Reset the barrier.
      comm->intraBarrier[comm->intraPhase^1] = 0;
    }
    done = __sync_bool_compare_and_swap(ptr, val, val+1);
    val++;
  }
  return ncclSuccess;
}
ncclResult_t ncclCpuBarrierWait(ncclComm_t comm) {
  if (comm->nRanks == 1) return ncclSuccess;
  // We can't print the CG mode before the first barrier happened.
  if (comm->rank == 0 && *comm->intraCGMode & 0x10) {
    *comm->intraCGMode ^= 0x10;
    INFO("Launch mode %s%s", comm->launchMode == ncclComm::GROUP ? "Group" : "Parallel", *comm->intraCGMode ? "/CGMD" : "" );
  }
  volatile int* ptr = (volatile int*)(comm->intraBarrier+comm->intraPhase);
  while (*ptr < comm->intraRanks) pthread_yield();
  comm->intraPhase ^= 1;
  if (comm->launchMode == ncclComm::GROUP) {
    CUDACHECK(hipEventRecord(comm->doneEvent, comm->ncclStream));
    CUDACHECK(hipStreamWaitEvent(comm->userStream, comm->doneEvent, 0));
  } else {
    struct hipLaunchParams *params = comm->intraParams+comm->intraRank;
    CUDACHECK(hipLaunchKernel(reinterpret_cast<const void*>(params->func), params->gridDim, params->blockDim, params->args, params->sharedMem, comm->userStream));
    CUDACHECK(hipEventRecord(comm->doneEvent, comm->userStream));
    comm->ncclStream = comm->userStream;
  }
  NCCLCHECK(transportStartProxies(comm));
  return ncclSuccess;
}

ncclResult_t ncclEnqueueCheck(ncclFunc_t func, const char* primName, const void* sendbuff, 
    void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root,
    ncclComm_t comm, hipStream_t stream) {
  // Launch asynchronously if needed
  if (ncclAsyncMode()) {
    if (ncclChecks) {
      int savedDev;
      CUDACHECK(hipGetDevice(&savedDev));
      CUDACHECK(hipSetDevice(comm->cudaDev));
      // Check arguments
      ncclResult_t ret = ArgsCheck(sendbuff, recvbuff, count, type, op, root, comm, primName);
      NCCLCHECK(ncclAsyncErrCheck(ret));
      CUDACHECK(hipSetDevice(savedDev));
    }
    NCCLCHECK(func(sendbuff, recvbuff, count, type, op, root, comm, stream));
    NCCLCHECK(ncclAsyncColl(comm));
    return ncclSuccess;
  } else {
    if (ncclChecks) NCCLCHECK(ArgsCheck(sendbuff, recvbuff, count, type, op, root, comm, primName));
    NCCLCHECK(func(sendbuff, recvbuff, count, type, op, root, comm, stream));
    NCCLCHECK(ncclCpuBarrierCheckin(comm));
    NCCLCHECK(ncclCpuBarrierWait(comm));
    return ncclSuccess;
  }
}
