/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" {
int __attribute__((weak)) ncclMpiEnabled() { fprintf(stderr, "No MPI\n"); return 0; }
int __attribute__((weak)) ncclMpiCudaSupport() { return 0; }
int __attribute__((weak)) ncclMpiCommRank(int *rank) { return 1; }
int __attribute__((weak)) ncclMpiGetTag(int *tag) { return 1; }
int __attribute__((weak)) ncclMpiSend(int rank, void* data, int size, int tag) { return 1; }
int __attribute__((weak)) ncclMpiRecv(int rank, void* data, int size, int tag) { return 1; }
}

