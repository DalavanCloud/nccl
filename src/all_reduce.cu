#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <assert.h>

#include "core.h"
#include "enqueue.h"
#include "primitives.h"


__device__ inline void getSliceSizeAndOffset(int *size, int *offset, int slice,
    int numSlices, int numBigSlices, int numSmallSlices, int bigSliceN,
    int smallSliceN, int lastSliceN) {
  if (slice < numBigSlices) {
    *size = bigSliceN;
    *offset = slice * bigSliceN;
  } else {
    *size = (slice < numBigSlices + numSmallSlices) ? smallSliceN
        : ((slice == numSlices - 1) ? lastSliceN : 0);
    *offset = numBigSlices * bigSliceN + (slice - numBigSlices) * smallSliceN;
  }
}

template <int THREADS, typename T> __device__ __forceinline__
void LoadRing(const DevRing<char>* src, DevRing<T>* dst) {
  enum { NUM_WORDS = sizeof(DevRing<char>) / sizeof(long long) };
  static_assert(sizeof(DevRing<char>) % sizeof(long long) == 0, "Bad alignment");
  static_assert(THREADS >= NUM_WORDS, "Not enough threads to load DevRing");
  static_assert(sizeof(DevRing<char>) == sizeof(DevRing<T>), "DevRing size mismatch");
  long long* lldst = reinterpret_cast<long long*>(dst);
  const long long* llsrc = reinterpret_cast<const long long*>(src);
  if (threadIdx.x < NUM_WORDS) {
    lldst[threadIdx.x] = llsrc[threadIdx.x];
  }
}

template<typename T>
struct AllReduceKernelArgs {
  // general parameters
  int NumGPUs;
  int N;
  int opIndex;
  volatile int * __restrict__ opCounter;
  int * __restrict__ doneCount;
  bool pushrecv;

  // some pre-computed sizes
  int SliceSize;
  int SliceOffset;
  int ChunkSize;
  int NumChunks;

  // local and remote input, output, and buffer
  const T * __restrict__ ThisInput;
  T * __restrict__ ThisOutput;

  DevRing<char>* rings;
};


template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const AllReduceKernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ T* sharedNextOutput;
  __shared__ DevRing<T> ring;
  bool pushrecv = args.pushrecv;

  LoadRing<THREADS>(args.rings+bid, &ring);
  __syncthreads();

  if (tid == 0) {
    if (pushrecv) {
      WaitFlag prevCommOp(ring.prevOpCounter);
      prevCommOp.wait(args.opIndex);

      *ring.sendPtrToPrev = (T*)args.ThisOutput;
      Wait([=] {
        return *ring.recvPtrFromNext != nullptr;
      });
      sharedNextOutput = *ring.recvPtrFromNext;
      *ring.recvPtrFromNext = nullptr;
    } else {
      WaitFlag nextCommOp(ring.nextOpCounter);
      nextCommOp.wait(args.opIndex);
    }
  }
  __syncthreads();

  WaitFlag thisChunkDone(ring.recvFlagFromNext);
  WaitFlag thisDataReady(ring.recvFlagFromPrev);
  PostFlag prevChunkDone(ring.sendFlagToPrev);
  PostFlag nextDataReady(ring.sendFlagToNext);

  typedef Primitives<THREADS, UNROLL, 2, T, FUNC> Prims;

  int step = 0;
  for (int chunk=bid; chunk<args.NumChunks; chunk+=gridDim.x) {
    // calculate slice size.  for all chunks except (possibly) the last one,
    // this will just be args.SliceSize. For the last one, it may be smaller
    int bigSliceN   = args.SliceSize;
    int smallSliceN = 0;
    int lastSliceN  = 0;
    int numSlices   = args.NumGPUs;
    int numBigSlices   = numSlices;
    int numSmallSlices = 0;

    // last chunk
    if ((chunk + 1 == args.NumChunks) && (args.N % args.ChunkSize > 0)) {
      if (!pushrecv) {
        thisChunkDone.wait(2*step); // TODO handle slice resize more elegantly.
      }
      CalcLastChunk<THREADS, UNROLL, T>(&bigSliceN, &smallSliceN, &lastSliceN,
          &numSlices, &numBigSlices, &numSmallSlices, args.N, args.NumChunks,
          args.ChunkSize);
    }

    // this offset is only applied to Data pointers, not to Buffer pointers,
    // since we only have one buffer per chunk
    int chunkOffset = chunk * args.ChunkSize;

    // Compute pointers
    const T * __restrict__ thisInput = args.ThisInput  + chunkOffset;
    T * __restrict__ thisOutput =  args.ThisOutput + chunkOffset;
    T * __restrict__ prevInput = ring.recvBuffer;
    T * __restrict__ nextOutput =  ring.sendBuffer;

    /////////////// begin AllReduce steps ///////////////

    // step 0: push data to next GPU
    int slice = ring.userRank[args.NumGPUs-1];
    int offset;
    int sliceSize;
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    if (pushrecv) {
      Prims::Copy(
          thisInput  + offset,
          nextOutput + offset,
          sliceSize,
          step++, nextDataReady);
    } else {
      Prims::Copy(
          thisInput  + offset,
          nextOutput + offset,
          sliceSize,
          step++, thisChunkDone, nextDataReady);
    }

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<args.NumGPUs; ++j) {
      slice = ring.userRank[args.NumGPUs-j];
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      Prims::Reduce(
          prevInput  + offset,
          thisInput  + offset,
          nextOutput + offset,
          sliceSize,
          step++, thisDataReady, nextDataReady);
    }

    if (pushrecv) nextOutput = sharedNextOutput + chunkOffset;

    // step k - 1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = ring.userRank[0];
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    Prims::ReduceCopy(
        prevInput  + offset,
        thisInput  + offset,
        nextOutput + offset,
        thisOutput + offset,
        sliceSize,
        step++, thisDataReady, nextDataReady);

    // k-2 steps: copy result to next GPU
    if (pushrecv) {
      for (int j=1; j<args.NumGPUs-1; ++j) {
	slice = ring.userRank[args.NumGPUs - j];
	getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
	    numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

	Prims::Copy(
	    thisOutput + offset,
	    nextOutput + offset,
	    sliceSize,
	    step++, thisDataReady, nextDataReady);
      }
    } else {
      for (int j=1; j<args.NumGPUs-1; ++j) {
	slice = ring.userRank[args.NumGPUs - j];
	getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
	    numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

	Prims::DoubleCopy(
	    prevInput  + offset,
	    nextOutput + offset,
	    thisOutput + offset,
	    sliceSize,
	    step++, thisDataReady, nextDataReady);
      }

      // Make final copy from buffer to dest.
      slice = ring.userRank[1];
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          prevInput  + offset,
          thisOutput + offset,
          sliceSize,
          step++, thisDataReady, prevChunkDone);
    }
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    if (pushrecv) {
      thisDataReady.wait(2*step); // wait to receive last data
    } else {
      thisChunkDone.wait(2*step); // wait for last flag update
      *ring.recvFlagFromNext = 0;
    }

    // Each CTA resets its own flags
    *ring.recvFlagFromPrev = 0;

    // Last CTA increments comm's operation counts
    if (atomicAdd(args.doneCount, 1) == gridDim.x-1) {
      *args.doneCount = 0;
      __threadfence_system(); // Technically need to ensure that cleared flags
                              // are visible before incrementing op counter.
      *args.opCounter = args.opIndex+1;
    }
  }
}

template<class FUNC, typename T>
ncclResult_t RingAllReduce(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  AllReduceKernelArgs<T> args;
  args.NumGPUs = comm->nRanks;
  args.N = count;
  args.opIndex = comm->opSched;
  args.opCounter = comm->opCounter;
  args.doneCount = comm->devMem->flags + MAXFLAGS-1;

  const int minSlice = 2 * NCCL_UNROLL_SIZE * sizeof(PackType) / sizeof(T);
  const int atomSize = minSlice * comm->nRanks;
  const int numAtoms = (count + atomSize-1) / atomSize;
  const int nRings = min(numAtoms, comm->nRings);
  const int maxAtomsPerChunk = comm->buffSizePerRing / (sizeof(T) * atomSize);
  assert (maxAtomsPerChunk > 1);

  if (numAtoms == nRings) {
    args.SliceSize = minSlice;
    args.ChunkSize = atomSize;
    args.NumChunks = numAtoms;
  } else { // numAtoms > nRings
    int minNumChunks = (numAtoms + maxAtomsPerChunk-1) / maxAtomsPerChunk;
    int targetChunks = ((minNumChunks + nRings-1) / nRings) * nRings;
    int atomsPerChunk = numAtoms / targetChunks;
    if (numAtoms % targetChunks > 1) {
      atomsPerChunk += 1;
      args.NumChunks = (numAtoms+atomsPerChunk-1) / atomsPerChunk;
    } else {
      args.NumChunks = targetChunks;
    }

    args.SliceSize = minSlice * atomsPerChunk;
    args.ChunkSize = atomSize * atomsPerChunk;
  }

  args.ThisInput = (const T*)sendbuff;
  args.ThisOutput = (T*)recvbuff;
  args.rings = comm->devRing;
  args.pushrecv = comm->globalMemSpace;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    LAUNCH_KERNEL(AllReduceKernel, args, stream, nRings, (comm->p2ptype == ncclComm::NVLINK));
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class AllReduce {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingAllReduce<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return enqueue<AllReduce>(sendbuff, recvbuff, count, datatype, op, 0, comm, stream);
}

