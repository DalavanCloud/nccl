#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <assert.h>

#include "core.h"
#include "enqueue.h"
#include "crc32.h"

#include "syncfunc.h"
#include "primstep.h"


#define UNROLL_SIZE     (UNROLL_COUNT * NUM_THREADS)


__device__ inline void getSliceSizeAndOffset(int *size, int *offset, int slice,
    int numSlices, int numBigSlices, int numSmallSlices, int bigSliceN,
    int smallSliceN, int lastSliceN) {
  if (slice < numBigSlices) {
    *size = bigSliceN;
    *offset = slice * bigSliceN;
  } else {
    *size = (slice < numBigSlices + numSmallSlices) ? smallSliceN
        : ((slice == numSlices - 1) ? lastSliceN : 0);
    *offset = numBigSlices * bigSliceN + (slice - numBigSlices) * smallSliceN;
  }
}


template<typename T>
struct AllReduceRingArgs {
  int ThisId;

  T ** ThisPtrToNextOutput;
  T ** PrevPtrToThisOutput;
  volatile int* __restrict__ NextOpCounter;
  volatile int* __restrict__ PrevOpCounter;

  volatile T * __restrict__ ThisBuffer;
  volatile T * __restrict__ NextBuffer;

  // local and remote flags
  volatile int * __restrict__ ThisNewDataAvailableFlag;
  volatile int * __restrict__ NextNewDataAvailableFlag;
  volatile int * __restrict__ ThisChunkDoneFlag;
  volatile int * __restrict__ PrevChunkDoneFlag;
};

template<typename T>
struct AllReduceKernelArgs {
  // general parameters
  int NumGPUs;
  int N;
  int opIndex;
  volatile int * __restrict__ opCounter;
  int * __restrict__ doneCount;

  // some pre-computed sizes
  int SliceSize;
  int ChunkSize;
  int NumChunks;

  // local and remote input, output, and buffer
  const T * __restrict__ ThisInput;
  volatile T * __restrict__ ThisOutput;

  AllReduceRingArgs<T> rings[MAXRINGS];
};

template<class FUNC, bool PUSHRECV, typename T>
__launch_bounds__(NUM_THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const AllReduceKernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ volatile void * nextOutput;
  __shared__ AllReduceRingArgs<T> ring;

  if (tid == 0) {
    ring = args.rings[bid];

    if (PUSHRECV) {
      auto prevCommOp = PackSyncFlags(ring.PrevOpCounter);
      prevCommOp.wait(args.opIndex);

      *((T * volatile *)ring.PrevPtrToThisOutput) = (T*)args.ThisOutput;
      Wait([=] {
        return *((T * volatile *)ring.ThisPtrToNextOutput) != nullptr;
      });
      nextOutput =
        *((volatile void * volatile *)ring.ThisPtrToNextOutput);
      *ring.ThisPtrToNextOutput = nullptr;
    } else {
      auto nextCommOp = PackSyncFlags(ring.NextOpCounter);
      nextCommOp.wait(args.opIndex);
    }
  }

  __syncthreads();

  auto dataForNext = PackSyncFlags(ring.NextNewDataAvailableFlag);
  auto dataForMe   = PackSyncFlags(ring.ThisNewDataAvailableFlag);
  auto buffForPrev = PackSyncFlags(ring.PrevChunkDoneFlag);
  auto buffForMe   = PackSyncFlags(ring.ThisChunkDoneFlag);

  int step = 0;
  for (int chunk=bid; chunk<args.NumChunks; chunk+=gridDim.x) {
    // calculate slice size.  for all chunks except (possibly) the last one,
    // this will just be args.SliceSize. For the last one, it may be smaller
    int bigSliceN   = args.SliceSize;
    int smallSliceN = 0;
    int lastSliceN  = 0;
    int numSlices   = args.NumGPUs;
    int numBigSlices   = numSlices;
    int numSmallSlices = 0;

    // last chunk
    if ((chunk + 1 == args.NumChunks) && (args.N % args.ChunkSize > 0))
      CalcLastChunk<NUM_THREADS, UNROLL_COUNT, T>(&bigSliceN, &smallSliceN, &lastSliceN,
          &numSlices, &numBigSlices, &numSmallSlices, args.N, args.NumChunks,
          args.ChunkSize);

    // this offset is only applied to Data pointers, not to Buffer pointers,
    // since we only have one buffer per chunk
    int chunkOffset = chunk * args.ChunkSize;

    /////////////// begin AllReduce steps ///////////////

    // step 0: push data to next GPU
    int slice = ring.ThisId;
    int offset;
    int sliceSize;
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    if (PUSHRECV) {
      CopyStep(step, 2, NOSYNC(),
          args.ThisInput + chunkOffset + offset,
          ring.NextBuffer + offset,
          sliceSize, dataForNext);
    } else {
      CopyStep(step, 2, buffForMe,
          args.ThisInput + chunkOffset + offset,
          ring.NextBuffer + offset,
          sliceSize, dataForNext);
    }
    ++ step;

    // steps j with 1 <= j < k - 1, where k = number of GPUs:
    // reduce and copy to next GPU
    for (int j=1; j<args.NumGPUs-1; ++j) {
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      ReduceStep<FUNC>(step, 2, dataForMe,
          ring.ThisBuffer + offset,
          args.ThisInput + chunkOffset + offset,
          ring.NextBuffer + offset,
          sliceSize, dataForNext);
      ++ step;
    }

    // step k - 1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    if (PUSHRECV) {
      ReduceCopyStep<FUNC>(step, 2, dataForMe,
          ring.ThisBuffer + offset,
          args.ThisInput + chunkOffset + offset,
          (volatile T *)nextOutput + chunkOffset + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize, dataForNext);
    } else {
      ReduceCopyStep<FUNC>(step, 2, dataForMe,
          ring.ThisBuffer + offset,
          args.ThisInput + chunkOffset + offset,
          ring.NextBuffer + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize, dataForNext);
    }
    ++ step;

    // steps j with k <= j < 2*k-2: copy result to next GPU
    for (int j=1; j<args.NumGPUs-1; ++j) {
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      if( PUSHRECV ) {
        CopyStep(step, 2, dataForMe,
            args.ThisOutput + chunkOffset + offset,
            (volatile T *)nextOutput + chunkOffset + offset,
            sliceSize, dataForNext);
      } else {
        DoubleCopyStep(step, 2, dataForMe,
            ring.ThisBuffer + offset,
            ring.NextBuffer + offset,
            args.ThisOutput + chunkOffset + offset,
            sliceSize, dataForNext);
      }
      ++ step;
    }

    if (!PUSHRECV) {
      // Make final copy from buffer to dest.
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      // Here we need to copy from buffer to this output.
      CopyStep(step, 2, dataForMe,
          ring.ThisBuffer + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize, buffForPrev);
      ++ step;
    }
  }

  // TODO: Wrap steps in object
  // TODO: put step index in object
  // TODO: add synchronize method?
  // TODO: add reset method to wait flags
  //       NOT for post flags

  // wait for the last data to be pushed to us
  if (tid == 0) {
    if (PUSHRECV) {
      dataForMe.wait(step * 4);
    }

    // Each CTA resets its own flags
    *ring.ThisNewDataAvailableFlag = 0;
    if(!PUSHRECV) {
      buffForMe.wait(step*4);
      *ring.ThisChunkDoneFlag = 0;
    }

    // Last CTA increments comm's operation counts
    if (atomicAdd(args.doneCount, 1) == gridDim.x-1) {
      *args.doneCount = 0;
      __threadfence_system(); // Technically need to ensure that cleared flags
                              // are visible before incrementing op counter.
      *args.opCounter = args.opIndex+1;
    }
  }
}

template<class FUNC, typename T>
ncclResult_t ncclAllReduceWithTypeAndFunc(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  AllReduceKernelArgs<T> args;
  args.NumGPUs = comm->nDev;
  args.N = count;
  args.opIndex = comm->opSched;
  args.opCounter = comm->opCounter;
  args.doneCount = comm->devMem->flags + MAXFLAGS-1;

  const int minSlice = 2 * UNROLL_SIZE * sizeof(PackType) / sizeof(T);
  const int atomSize = minSlice * comm->nDev;
  const int numAtoms = (count + atomSize-1) / atomSize;
  const int nRings = min(numAtoms, comm->nRings);
  const int maxAtomsPerChunk = (comm->buffSize / (nRings * sizeof(T) * atomSize));
  assert (maxAtomsPerChunk > 1);
  const int bufferOffset = maxAtomsPerChunk * atomSize;

  if (numAtoms == nRings) {
    args.SliceSize = minSlice;
    args.ChunkSize = atomSize;
    args.NumChunks = numAtoms;
  } else { // numAtoms > nRings
    int minNumChunks = (numAtoms + maxAtomsPerChunk-1) / maxAtomsPerChunk;
    int targetChunks = ((minNumChunks + nRings-1) / nRings) * nRings;
    int atomsPerChunk = numAtoms / targetChunks;
    if (numAtoms % targetChunks > 1) {
      atomsPerChunk += 1;
      args.NumChunks = (numAtoms+atomsPerChunk-1) / atomsPerChunk;
    } else {
      args.NumChunks = targetChunks;
    }

    args.SliceSize = minSlice * atomsPerChunk;
    args.ChunkSize = atomSize * atomsPerChunk;
  }

  args.ThisInput = (const T*)sendbuff;
  args.ThisOutput = (volatile T*)recvbuff;

  for(int r=0; r<nRings; ++r) {
    AllReduceRingArgs<T>& ring = args.rings[r];
    int index = comm->ringIdx[r];
    int nextId = comm->ncclFromRing[r][(index + 1) % comm->nDev];
    int prevId = comm->ncclFromRing[r][(index + comm->nDev - 1) % comm->nDev];

    ring.ThisId = index;
    ring.ThisPtrToNextOutput = (T**)&(comm->ptrs[nextId].local->recvPtrs[r]);
    ring.PrevPtrToThisOutput = (T**)&(comm->ptrs[prevId].remote->recvPtrs[r]);
    ring.NextOpCounter = comm->ptrs[nextId].opCounter;
    ring.PrevOpCounter = comm->ptrs[prevId].opCounter;
    ring.ThisBuffer = (volatile T*)comm->ptrs[prevId].local->buff + r*bufferOffset;
    ring.NextBuffer = (volatile T*)comm->ptrs[nextId].remote->buff + r*bufferOffset;
    ring.ThisNewDataAvailableFlag = comm->ptrs[prevId].local->flags + r;
    ring.NextNewDataAvailableFlag = comm->ptrs[nextId].remote->flags + r;
    ring.ThisChunkDoneFlag = comm->ptrs[nextId].local->flags + nRings + r;
    ring.PrevChunkDoneFlag = comm->ptrs[prevId].remote->flags + nRings + r;
  }

  // print CRC checksum of input
  int myRank;
  if (ncclPrintCRCs) {
    myRank = comm->userFromRing[0][comm->ringIdx[0]];
    printCRCDev((unsigned char*)sendbuff, count*sizeof(T), myRank, stream);
  }

  dim3 grid(nRings, 1, 1);
  dim3 block(NUM_THREADS+1, 1, 1);
  void* argptrs[] = {&args};
  if( comm->useRemoteRecv ) {
    CUDACHECK(hipLaunchKernel(
        (void*)AllReduceKernel<FUNC, true, T>,
        grid, block, argptrs, 0, stream));
  } else {
    CUDACHECK(hipLaunchKernel(
        (void*)AllReduceKernel<FUNC, false, T>,
        grid, block, argptrs, 0, stream));
  }

  // print CRC checksum of output
  if (ncclPrintCRCs) {
    printCRCDev((unsigned char*)recvbuff, count*sizeof(T), myRank, stream);
  }

  return ncclSuccess;
}


template<typename T>
ncclResult_t ncclAllReduceWithType(const void* sendbuff,
    void* recvbuff, int count, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  switch (op) {
  case ncclSum:
    return ncclAllReduceWithTypeAndFunc<FuncSum<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclProd:
    return ncclAllReduceWithTypeAndFunc<FuncProd<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclMax:
    return ncclAllReduceWithTypeAndFunc<FuncMax<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclMin:
    return ncclAllReduceWithTypeAndFunc<FuncMin<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  }
  return ncclInvalidOperation;
}

class AllReduceFunctor {
public:
  ncclResult_t operator()(const void* sendbuff, void* recvbuff,
      int count, ncclDataType_t datatype, ncclRedOp_t op, int /*root*/,
      ncclComm* comm, hipStream_t stream) {

    switch (datatype) {
    case ncclChar:
      return ncclAllReduceWithType<char>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclInt:
      return ncclAllReduceWithType<int>(sendbuff, recvbuff, count, op,
          comm, stream);
#ifdef CUDA_HAS_HALF
    case ncclHalf:
      return ncclAllReduceWithType<half>(sendbuff, recvbuff, count, op,
          comm, stream);
#endif
    case ncclFloat:
      return ncclAllReduceWithType<float>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclDouble:
      return ncclAllReduceWithType<double>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclInt64:
      return ncclAllReduceWithType<long long>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclUint64:
      return ncclAllReduceWithType<unsigned long long int>(sendbuff, recvbuff, count, op,
          comm, stream);
    }

    return ncclInvalidType;
  }
};

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return enqueue(AllReduceFunctor(), sendbuff, recvbuff, count, datatype, op, 0,
      comm, stream);
}

