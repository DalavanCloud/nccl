#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ T* sharedNextOutput;
  struct ncclComm* comm = args.comm;
  struct ncclRing* ring = comm->rings+bid;
  int prevdirect = ring->sendrecv.recv.conn.direct;
  int nextdirect = ring->sendrecv.send.conn.direct;

  if (tid == 0) {
    // Wait for prev and next to be ready
    Wait([=] {
        return *ring->sendrecv.recv.conn.head == 0;
    });
    Wait([=] {
        return *ring->sendrecv.send.conn.tail == 0;
    });
    
    if (prevdirect) {
      *ring->sendrecv.recv.conn.ptrExchange = (T*)args.ThisOutput;
    }
    if (nextdirect) {
      Wait([=] {
        return *(ring->sendrecv.send.conn.ptrExchange) != nullptr;
      });
      sharedNextOutput = (T*)*ring->sendrecv.send.conn.ptrExchange;
      *ring->sendrecv.send.conn.ptrExchange = nullptr;
    }
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring->sendrecv.send.conn.head, -NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring->sendrecv.recv.conn.tail, -1*NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring->sendrecv.recv.conn.head, -1*NUM_SUBSTEPS);
  PostFlag postReadyToNext(ring->sendrecv.send.conn.tail, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = comm->nRanks;
  const int buffSize = ring->buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput = args.ThisOutput;
  T * __restrict__ prevInput = (T*)ring->sendrecv.recv.conn.buff;
  T * __restrict__ nextOutput = (T*)ring->sendrecv.send.conn.buff;

  for (int gridOffset = 0; gridOffset < size; gridOffset += gridDim.x*nranks*sliceSize) {
    /////////////// begin AllReduce steps ///////////////
    int offset;
    int maxOffset;
    int slice;
    int chunkSize = min(sliceSize, DIVUP(size-gridOffset,nranks*gridDim.x));
    ALIGN_SIZE(chunkSize, THREADS*UNROLL);
    int chunkOffset = gridOffset + bid*nranks*chunkSize;

    // step 0: push data to next GPU
    slice = ring->userRanks[nranks-1];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::Copy(
        thisInput  + offset,
        nextOutput + noffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      slice = ring->userRanks[nranks-j];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      Prims::Reduce(
          prevInput  + poffset,
          thisInput  + offset,
          nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = ring->userRanks[0];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::ReduceCopy(
        prevInput  + poffset,
        thisInput  + offset,
        nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
        thisOutput + offset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP;

    // k-2 steps: copy to next GPU
    if (prevdirect) {
      for (int j=1; j<nranks-1; ++j) {
        slice = ring->userRanks[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::Copy(
            thisOutput + offset,
	    nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }
    } else {
      for (int j=1; j<nranks-1; ++j) {
        slice = ring->userRanks[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::DoubleCopy(
            prevInput + poffset,
            thisOutput + offset,
	    nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }

      // Make final copy from buffer to dest.
      slice = ring->userRanks[1];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          prevInput + poffset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    // Wait for last update from next then reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
    *ring->sendrecv.send.conn.head = 0;

    // Wait for last update from prev then reset the flag
    waitReadyFromPrev.wait(NUM_SUBSTEPS*(step+1));
    *ring->sendrecv.recv.conn.tail = 0;
  }
}

#define PCIE_THREADS 512
#define NVLINK_THREADS 128
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingAllReduce(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, 0, count, comm);
    if (comm->p2ptype == ncclComm::NVLINK) {
      LAUNCH_KERNEL(AllReduceKernel, NVLINK_THREADS, UNROLL, FUNC, T, args, stream);
    } else {
      LAUNCH_KERNEL(AllReduceKernel, PCIE_THREADS, UNROLL, FUNC, T, args, stream);
    }
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class AllReduce {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingAllReduce<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return enqueue<AllReduce>(sendbuff, recvbuff, count, datatype, op, 0, comm, stream);
}

