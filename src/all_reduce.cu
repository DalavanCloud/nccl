#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <assert.h>

#include "core.h"
#include "enqueue.h"
#include "crc32.h"

#include "primitives.h"


__device__ inline void getSliceSizeAndOffset(int *size, int *offset, int slice,
    int numSlices, int numBigSlices, int numSmallSlices, int bigSliceN,
    int smallSliceN, int lastSliceN) {
  if (slice < numBigSlices) {
    *size = bigSliceN;
    *offset = slice * bigSliceN;
  } else {
    *size = (slice < numBigSlices + numSmallSlices) ? smallSliceN
        : ((slice == numSlices - 1) ? lastSliceN : 0);
    *offset = numBigSlices * bigSliceN + (slice - numBigSlices) * smallSliceN;
  }
}


template<typename T>
struct AllReduceRingArgs {
  int ThisId;

  T ** ThisPtrToNextOutput;
  T ** PrevPtrToThisOutput;
  volatile int* __restrict__ NextOpCounter;
  volatile int* __restrict__ PrevOpCounter;

  volatile T * __restrict__ ThisBuffer;
  volatile T * __restrict__ NextBuffer;

  // local and remote flags
  volatile int * __restrict__ ThisNewDataAvailableFlag;
  volatile int * __restrict__ NextNewDataAvailableFlag;
  volatile int * __restrict__ ThisChunkDoneFlag;
  volatile int * __restrict__ PrevChunkDoneFlag;
};

template<typename T>
struct AllReduceKernelArgs {
  // general parameters
  int NumGPUs;
  int N;
  int opIndex;
  volatile int * __restrict__ opCounter;
  int * __restrict__ doneCount;

  // some pre-computed sizes
  int SliceSize;
  int SliceOffset;
  int ChunkSize;
  int NumChunks;

  // local and remote input, output, and buffer
  const T * __restrict__ ThisInput;
  volatile T * __restrict__ ThisOutput;

  AllReduceRingArgs<T> rings[MAXRINGS];
};

template<int THREADS, int UNROLL, class FUNC, bool PUSHRECV, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const AllReduceKernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ volatile T* nextOutput;
  __shared__ AllReduceRingArgs<T> ring;

  if (tid == 0) {
    ring = args.rings[bid];

    if (PUSHRECV) {
      WaitFlag prevCommOp(ring.PrevOpCounter);
      prevCommOp.wait(args.opIndex);

      *((T * volatile *)ring.PrevPtrToThisOutput) = (T*)args.ThisOutput;
      Wait([=] {
        return *((T * volatile *)ring.ThisPtrToNextOutput) != nullptr;
      });
      nextOutput =
        *((volatile T * volatile *)ring.ThisPtrToNextOutput);
      *ring.ThisPtrToNextOutput = nullptr;
    } else {
      WaitFlag nextCommOp(ring.NextOpCounter);
      nextCommOp.wait(args.opIndex);
    }
  }
  __syncthreads();

  WaitFlag thisChunkDone(ring.ThisChunkDoneFlag);
  WaitFlag thisDataReady(ring.ThisNewDataAvailableFlag);
  PostFlag prevChunkDone(ring.PrevChunkDoneFlag);
  PostFlag nextDataReady(ring.NextNewDataAvailableFlag);

  typedef Primitives<THREADS, UNROLL, 2, T, FUNC> Prims;

  int step = 0;
  for (int chunk=bid; chunk<args.NumChunks; chunk+=gridDim.x) {
    // calculate slice size.  for all chunks except (possibly) the last one,
    // this will just be args.SliceSize. For the last one, it may be smaller
    int bigSliceN   = args.SliceSize;
    int smallSliceN = 0;
    int lastSliceN  = 0;
    int numSlices   = args.NumGPUs;
    int numBigSlices   = numSlices;
    int numSmallSlices = 0;

    // last chunk
    if ((chunk + 1 == args.NumChunks) && (args.N % args.ChunkSize > 0)) {
      if (!PUSHRECV) {
        thisChunkDone.wait(2*step); // TODO handle slice resize more elegantly.
      }
      CalcLastChunk<THREADS, UNROLL, T>(&bigSliceN, &smallSliceN, &lastSliceN,
          &numSlices, &numBigSlices, &numSmallSlices, args.N, args.NumChunks,
          args.ChunkSize);
    }

    // this offset is only applied to Data pointers, not to Buffer pointers,
    // since we only have one buffer per chunk
    int chunkOffset = chunk * args.ChunkSize;

    /////////////// begin AllReduce steps ///////////////

    // step 0: push data to next GPU
    int slice = ring.ThisId;
    int offset;
    int sliceSize;
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    if (PUSHRECV) {
      Prims::Copy(
          args.ThisInput  + chunkOffset + offset,
          ring.NextBuffer               + offset,
          sliceSize,
          step++, nextDataReady);
    } else {
      Prims::Copy(
          args.ThisInput  + chunkOffset + offset,
          ring.NextBuffer               + offset,
          sliceSize,
          step++, thisChunkDone, nextDataReady);
    }

    // steps j with 1 <= j < k - 1, where k = number of GPUs:
    // reduce and copy to next GPU
    for (int j=1; j<args.NumGPUs-1; ++j) {
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      Prims::Reduce(
          ring.ThisBuffer               + offset,
          args.ThisInput  + chunkOffset + offset,
          ring.NextBuffer               + offset,
          sliceSize,
          step++, thisDataReady, nextDataReady);
    }

    // step k - 1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
    getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
        numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

    if (PUSHRECV) {
      Prims::ReduceCopy(
          ring.ThisBuffer               + offset,
          args.ThisInput  + chunkOffset + offset,
          nextOutput      + chunkOffset + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize,
          step++, thisDataReady, nextDataReady);
    } else {
      Prims::ReduceCopy(
          ring.ThisBuffer               + offset,
          args.ThisInput  + chunkOffset + offset,
          ring.NextBuffer               + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize,
          step++, thisDataReady, nextDataReady);
    }

    // steps j with k <= j < 2*k-2: copy result to next GPU
    for (int j=1; j<args.NumGPUs-1; ++j) {
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      if (PUSHRECV) {
        Prims::Copy(
            args.ThisOutput + chunkOffset + offset,
            nextOutput      + chunkOffset + offset,
            sliceSize,
            step++, thisDataReady, nextDataReady);
      } else {
        Prims::DoubleCopy(
            ring.ThisBuffer               + offset,
            ring.NextBuffer               + offset,
            args.ThisOutput + chunkOffset + offset,
            sliceSize,
            step++, thisDataReady, nextDataReady);
      }
    }

    if (!PUSHRECV) {
      // Make final copy from buffer to dest.
      slice = (args.NumGPUs + slice - 1) % args.NumGPUs;
      getSliceSizeAndOffset(&sliceSize, &offset, slice, numSlices,
          numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          ring.ThisBuffer               + offset,
          args.ThisOutput + chunkOffset + offset,
          sliceSize,
          step++, thisDataReady, prevChunkDone);
    }
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    if (PUSHRECV) {
      thisDataReady.wait(2*step); // wait to receive last data
    } else {
      thisChunkDone.wait(2*step); // wait for last flag update
      *ring.ThisChunkDoneFlag = 0;
    }

    // Each CTA resets its own flags
    *ring.ThisNewDataAvailableFlag = 0;

    // Last CTA increments comm's operation counts
    if (atomicAdd(args.doneCount, 1) == gridDim.x-1) {
      *args.doneCount = 0;
      __threadfence_system(); // Technically need to ensure that cleared flags
                              // are visible before incrementing op counter.
      *args.opCounter = args.opIndex+1;
    }
  }
}

template<class FUNC, typename T>
ncclResult_t ncclAllReduceWithTypeAndFunc(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  enum {THREADS = 256};
  enum {UNROLL = 8};
  enum {UNROLL_SIZE = THREADS*UNROLL};

  AllReduceKernelArgs<T> args;
  args.NumGPUs = comm->nDev;
  args.N = count;
  args.opIndex = comm->opSched;
  args.opCounter = comm->opCounter;
  args.doneCount = comm->devMem->flags + MAXFLAGS-1;

  // START
  //const int minChunkSize = comm->nDev * 2 * UNROLL_SIZE * sizeof(PackType) / sizeof(T);
  //const int maxNumChunks = count / minChunkSize;
  //const int nRings = std::min(comm->nRings, maxNumChunks);
  //const int 
  //N <= numGpus * chunksPerGpu * chunkSize <= N+numGpus*minChunkSize-1
  // STOP

  const int minSlice = 2 * UNROLL_SIZE * sizeof(PackType) / sizeof(T);
  const int atomSize = minSlice * comm->nDev;
  const int numAtoms = (count + atomSize-1) / atomSize;
  const int nRings = min(numAtoms, comm->nRings);
  const int maxAtomsPerChunk = (comm->buffSize / (nRings * sizeof(T) * atomSize));
  assert (maxAtomsPerChunk > 1);
  const int bufferOffset = maxAtomsPerChunk * atomSize;

  if (numAtoms == nRings) {
    args.SliceSize = minSlice;
    args.ChunkSize = atomSize;
    args.NumChunks = numAtoms;
  } else { // numAtoms > nRings
    int minNumChunks = (numAtoms + maxAtomsPerChunk-1) / maxAtomsPerChunk;
    int targetChunks = ((minNumChunks + nRings-1) / nRings) * nRings;
    int atomsPerChunk = numAtoms / targetChunks;
    if (numAtoms % targetChunks > 1) {
      atomsPerChunk += 1;
      args.NumChunks = (numAtoms+atomsPerChunk-1) / atomsPerChunk;
    } else {
      args.NumChunks = targetChunks;
    }

    args.SliceSize = minSlice * atomsPerChunk;
    args.ChunkSize = atomSize * atomsPerChunk;
  }

  args.ThisInput = (const T*)sendbuff;
  args.ThisOutput = (volatile T*)recvbuff;

  for(int r=0; r<nRings; ++r) {
    AllReduceRingArgs<T>& ring = args.rings[r];
    int index = comm->ringIdx[r];
    int nextId = comm->ncclFromRing[r][(index + 1) % comm->nDev];
    int prevId = comm->ncclFromRing[r][(index + comm->nDev - 1) % comm->nDev];

    ring.ThisId = index;
    ring.ThisPtrToNextOutput = (T**)&(comm->ptrs[nextId].local->recvPtrs[r]);
    ring.PrevPtrToThisOutput = (T**)&(comm->ptrs[prevId].remote->recvPtrs[r]);
    ring.NextOpCounter = comm->ptrs[nextId].opCounter;
    ring.PrevOpCounter = comm->ptrs[prevId].opCounter;
    ring.ThisBuffer = (volatile T*)comm->ptrs[prevId].local->buff + r*bufferOffset;
    ring.NextBuffer = (volatile T*)comm->ptrs[nextId].remote->buff + r*bufferOffset;
    ring.ThisNewDataAvailableFlag = comm->ptrs[prevId].local->flags + r;
    ring.NextNewDataAvailableFlag = comm->ptrs[nextId].remote->flags + r;
    ring.ThisChunkDoneFlag = comm->ptrs[nextId].local->flags + nRings + r;
    ring.PrevChunkDoneFlag = comm->ptrs[prevId].remote->flags + nRings + r;
  }

  // print CRC checksum of input
  int myRank;
  if (ncclPrintCRCs) {
    myRank = comm->userFromRing[0][comm->ringIdx[0]];
    printCRCDev((unsigned char*)sendbuff, count*sizeof(T), myRank, stream);
  }

  dim3 grid(nRings, 1, 1);
  dim3 block(THREADS+1, 1, 1);
  void* argptrs[] = {&args};
  if( comm->useRemoteRecv ) {
    CUDACHECK(hipLaunchKernel(
        (void*)AllReduceKernel<THREADS, UNROLL, FUNC, true, T>,
        grid, block, argptrs, 0, stream));
  } else {
    CUDACHECK(hipLaunchKernel(
        (void*)AllReduceKernel<THREADS, UNROLL, FUNC, false, T>,
        grid, block, argptrs, 0, stream));
  }

  // print CRC checksum of output
  if (ncclPrintCRCs) {
    printCRCDev((unsigned char*)recvbuff, count*sizeof(T), myRank, stream);
  }

  return ncclSuccess;
}


template<typename T>
ncclResult_t ncclAllReduceWithType(const void* sendbuff,
    void* recvbuff, int count, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  switch (op) {
  case ncclSum:
    return ncclAllReduceWithTypeAndFunc<FuncSum<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclProd:
    return ncclAllReduceWithTypeAndFunc<FuncProd<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclMax:
    return ncclAllReduceWithTypeAndFunc<FuncMax<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  case ncclMin:
    return ncclAllReduceWithTypeAndFunc<FuncMin<T>, T>(
        sendbuff, recvbuff, count, comm, stream);
  }
  return ncclInvalidOperation;
}

class AllReduceFunctor {
public:
  ncclResult_t operator()(const void* sendbuff, void* recvbuff,
      int count, ncclDataType_t datatype, ncclRedOp_t op, int /*root*/,
      ncclComm* comm, hipStream_t stream) {

    switch (datatype) {
    case ncclChar:
      return ncclAllReduceWithType<char>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclInt:
      return ncclAllReduceWithType<int>(sendbuff, recvbuff, count, op,
          comm, stream);
#ifdef CUDA_HAS_HALF
    case ncclHalf:
      return ncclAllReduceWithType<half>(sendbuff, recvbuff, count, op,
          comm, stream);
#endif
    case ncclFloat:
      return ncclAllReduceWithType<float>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclDouble:
      return ncclAllReduceWithType<double>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclInt64:
      return ncclAllReduceWithType<long long>(sendbuff, recvbuff, count, op,
          comm, stream);
    case ncclUint64:
      return ncclAllReduceWithType<unsigned long long int>(sendbuff, recvbuff, count, op,
          comm, stream);
    }

    return ncclInvalidType;
  }
};

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return enqueue(AllReduceFunctor(), sendbuff, recvbuff, count, datatype, op, 0,
      comm, stream);
}

