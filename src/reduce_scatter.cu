#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void ReduceScatterKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  struct ncclComm* comm = args.comm;
  struct ncclRing* ring = comm->rings+bid;

  if (tid == 0) {
    // Wait for prev and next to be ready
    Wait([=] {
        return *ring->recv.conn.head == 0;
    });
    Wait([=] {
        return *ring->send.conn.tail == 0;
    });
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring->send.conn.head, -NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring->recv.conn.tail, -1*NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring->recv.conn.head, -1*NUM_SUBSTEPS);
  PostFlag postReadyToNext(ring->send.conn.tail, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = comm->nRanks;
  const int buffSize = ring->buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput = args.ThisOutput;
  T * __restrict__ prevInput = (T*)ring->recv.conn.buff;
  T * __restrict__ nextOutput = (T*)ring->send.conn.buff;

  for (int chunkOffset = bid*sliceSize; chunkOffset < size; chunkOffset += gridDim.x*sliceSize) {
    /////////////// begin ReduceScatter steps ///////////////
    int offset;
    int maxOffset = size-chunkOffset;
    int rankDest;

    // step 0: push data to next GPU
    rankDest = ring->userRanks[nranks-1];
    offset = chunkOffset + rankDest * size;

    Prims::Copy(
        thisInput  + offset,
        nextOutput + noffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      rankDest = ring->userRanks[nranks-j];
      offset = chunkOffset + rankDest * size;

      Prims::Reduce(
          prevInput  + poffset,
          thisInput  + offset,
          nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    rankDest = ring->userRanks[0];
    offset = chunkOffset + rankDest * size;

    Prims::Reduce(
        prevInput  + poffset,
        thisInput  + offset,
        thisOutput + chunkOffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP;
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    // Wait for last update from next then reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
    *ring->send.conn.head = 0;

    // Wait for last update from prev then reset the flag
    waitReadyFromPrev.wait(NUM_SUBSTEPS*(step+1));
    *ring->recv.conn.tail = 0;
  }
}

#define PCIE_THREADS 512
#define NVLINK_THREADS 128
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingReduceScatter(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, 0, count, comm);
    if (comm->p2ptype == ncclComm::NVLINK) {
      LAUNCH_KERNEL(ReduceScatterKernel, NVLINK_THREADS, UNROLL, FUNC, T, args, stream);
    } else {
      LAUNCH_KERNEL(ReduceScatterKernel, PCIE_THREADS, UNROLL, FUNC, T, args, stream);
    }
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class ReduceScatter {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingReduceScatter<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclReduceScatter, const void* sendbuff, void* recvbuff, int recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream);
ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, int recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  return enqueue<ReduceScatter>(sendbuff, recvbuff, recvcount, datatype, op, 0, comm, stream);
}

