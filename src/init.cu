/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "nvmlwrap.h"
#include "topo.h"
#include "bootstrap.h"
#include "transport.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sched.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>

DebugLevel ncclDebugLevel;
int ncclPrintCRCs;

NCCL_API(ncclResult_t, ncclGetUniqueId, ncclUniqueId* out);
ncclResult_t ncclGetUniqueId(ncclUniqueId* out) {
  bootstrapGetUniqueId(out);
  if (out == NULL) {
    WARN("Error : no bootstrap available");
    return ncclInternalError;
  }
  return ncclSuccess;
}

static void initDebug() {
  const char* nccl_debug = getenv("NCCL_DEBUG");
  if (nccl_debug == NULL) {
    ncclDebugLevel = NONE;
  } else if (strcmp(nccl_debug, "VERSION") == 0) {
    ncclDebugLevel = VERSION;
  } else if (strcmp(nccl_debug, "WARN") == 0) {
    ncclDebugLevel = WARN;
  } else if (strcmp(nccl_debug, "INFO") == 0) {
    ncclDebugLevel = INFO;
    INFO("NCCL debug level set to INFO");
  } else if (strcmp(nccl_debug, "ABORT") == 0) {
    ncclDebugLevel = ABORT;
    INFO("NCCL debug level set to ABORT");
  }

  const char* nccl_crc = getenv("NCCL_CRC");
  if (nccl_crc != NULL && strcmp(nccl_crc, "PRINT")==0 ) {
    ncclPrintCRCs = 1;
  } else {
    ncclPrintCRCs = 0;
  }
}

static ncclResult_t commFree(ncclComm_t comm) {
  if (comm == NULL)
    return ncclSuccess;

  if (comm->doneEvent != NULL)
    CUDACHECK(hipEventDestroy(comm->doneEvent));

  free(comm);
  return ncclSuccess;
}

static ncclResult_t commAlloc(ncclComm_t* comret, int ndev, int rank) {
  if (ndev < 1) {
    WARN("invalid device count (%d) requested", ndev);
    return ncclUnsupportedDeviceCount;
  }
  if (rank >= ndev || rank < 0) {
    WARN("rank %d exceeds ndev=%d", rank, ndev);
    return ncclInvalidRank;
  }

  struct ncclComm* comm = (struct ncclComm*)malloc(sizeof(struct ncclComm));
  if (comm == NULL) {
    WARN("comm allocation failed");
    return ncclSystemError;
  }
  memset(comm, 0, sizeof(struct ncclComm));

  comm->rank = rank;
  comm->nRanks = ndev;
  hipGetDevice(&comm->cudaDev);

  if (hipEventCreateWithFlags(&comm->doneEvent, hipEventDisableTiming) != hipSuccess) {
    WARN("ncclComm on rank %d failed to create doneEvent", rank);
    commFree(comm);
    return ncclUnhandledCudaError;
  }

  *comret = comm;
  return ncclSuccess;
}

static ncclResult_t devCommSetup(ncclComm_t comm) {
  // Fully duplicate the comm on the device
  if (hipMalloc(&comm->devComm, sizeof(struct ncclComm)) != hipSuccess) {
    WARN("failed to allocated device comm");
    return ncclCudaMallocFailed;
  }
  // Copy the comm on the device
  if (hipMemcpy(comm->devComm, comm, sizeof(struct ncclComm), hipMemcpyHostToDevice) != hipSuccess) {
    WARN("failed to copy device comm");
    return ncclUnhandledCudaError;
  }
  return ncclSuccess;
}

static void showVersion() {
  static int shown = 0;
  if (shown == 0 && ncclDebugLevel >= VERSION) {
    printf("NCCL version %d.%d.%d compiled with CUDA %d.%d\n", NCCL_MAJOR, NCCL_MINOR, NCCL_PATCH, CUDA_MAJOR, CUDA_MINOR);
    fflush(stdout);
    shown = 1;
  }
}

static ncclResult_t fillInfo(struct ncclInfo* info, int rank) {
  for (int t=0; t<NTRANSPORTS; t++) {
    NCCLCHECK(ncclTransports[t].fillInfo(info->tinfo+t, rank));
  }
  return ncclSuccess;
}

template <int type>
static ncclResult_t selectTransport(struct ncclInfo* myInfo, struct ncclInfo* peerInfo, struct ncclConnect* connect, struct ncclTransport** transport, struct ncclRing* ring) {
  for (int t=0; t<NTRANSPORTS; t++) {
    struct ncclTransportComm* transportComm = type == 1 ? &ncclTransports[t].send : &ncclTransports[t].recv;
    int select = 0;
    NCCLCHECK(transportComm->setup(myInfo->tinfo+t, peerInfo->tinfo+t, connect, ring, &select));
    if (select == 1) {
      *transport = ncclTransports+t;
      return ncclSuccess;
    }
  }
  WARN("No transport found !");
  return ncclInternalError;
}

static ncclResult_t setupSendRecv(struct ncclRing* ring) {
  const char* str = getenv("NCCL_BUFFSIZE");
  int buffSize;
  if (str != NULL) {
    errno = 0;
    buffSize = strtol(str, NULL, 10);
    if (errno == ERANGE || buffSize == 0) {
      INFO("invalid NCCL_BUFFSIZE: %s, using default %lu",
          str, DEFAULT_BUFFER_SIZE_BYTES);
      buffSize = DEFAULT_BUFFER_SIZE_BYTES;
    }
  } else {
    buffSize = DEFAULT_BUFFER_SIZE_BYTES;
  }
  ring->buffSize = buffSize;
  const int size = ring->devMemSize = offsetof(struct ncclSendRecvMem, buff)+buffSize;
  struct ncclSendRecvMem* mem;
  CUDACHECK(hipMalloc(&mem, size));
  CUDACHECK(hipMemset(mem, 0, size));
  ring->devMem = mem;
  ring->recv.conn.buff = (char*)&mem->buff;
  ring->recv.conn.tail = &mem->tail;
  ring->recv.conn.direct = 0;
  ring->send.conn.head = &mem->head;
  ring->send.conn.direct = 0;
  return ncclSuccess;
}

static int getRings(int** rings, int nranks) {
  // TODO : something better !
  int *ptr = (int*)malloc(sizeof(int)*nranks);
  for (int i=0; i<nranks; i++)
    ptr[i] = i;
  *rings = ptr;
  return 1;
}

static ncclResult_t setupRing(struct ncclRing* ring, int ringid, int rank, int nranks, int* ringRanks, struct ncclInfo* allInfo, struct ncclConnect* connect) { 
  ring->id = ringid;
  // Reorganize ranks to start with rank.
  int shift;
  for (shift = 0; shift<nranks; shift++) {
    if (ringRanks[shift] == rank) {
      ring->rank = shift;
      break;
    }
  }
  for (int i=0; i<nranks; i++) {
    ring->userRanks[i] = ringRanks[(i+shift)%nranks];
  }
  int prev = ring->userRanks[nranks-1];
  int next = ring->userRanks[1];

  setupSendRecv(ring);
  NCCLCHECK(selectTransport<0>(allInfo+rank, allInfo+prev, connect+0, &ring->recv.transport, ring));
  NCCLCHECK(selectTransport<1>(allInfo+rank, allInfo+next, connect+1, &ring->send.transport, ring));
  NCCLCHECK(transportCreateProxy(0, ring));
  NCCLCHECK(transportCreateProxy(1, ring));
  return ncclSuccess;
}

static void swap(void* mem1, void* mem2, int size) {
  char tmp[size];
  memcpy(tmp, mem1, size); memcpy(mem1, mem2, size); memcpy(mem2, tmp, size);
}

static ncclResult_t initTransportsAll(struct ncclComm** comms, const int* devs, int nranks) {
  struct ncclInfo* allInfo = (struct ncclInfo*)malloc(sizeof(struct ncclInfo)*nranks);
  for (int rank=0; rank<nranks; rank++) {
    hipSetDevice(devs[rank]);
    fillInfo(allInfo+rank, rank);
  }
  
  int *rings;
  int nrings = getRings(&rings, nranks);

  for (int rank=0; rank<nranks; rank++)
    comms[rank]->nRings = nrings;

  for (int r=0; r<nrings; r++) {
    struct ncclConnect connect[2*nranks];
    int* ringRanks = rings+r*nranks;
    for (int rank=0; rank<nranks; rank++) {
      CUDACHECK(hipSetDevice(devs[rank]));
      struct ncclRing *ring = comms[rank]->rings+r;
      NCCLCHECK(setupRing(ring, r, rank, nranks, ringRanks, allInfo, connect+2*rank));
    }
    // RingExchange connect information
    for (int rank=0; rank<nranks; rank++) {
      // Swap rank->prev and prevRank->next
      struct ncclRing *ring = comms[rank]->rings+r;
      int prevRank = ring->userRanks[nranks-1];
      struct ncclConnect* prevRankNextConnect = connect+2*prevRank+1;
      struct ncclConnect* rankPrevConnect = connect+2*rank;
      swap(prevRankNextConnect, rankPrevConnect, sizeof(struct ncclConnect));
    }
    for (int rank=0; rank<nranks; rank++) {
      CUDACHECK(hipSetDevice(devs[rank]));
      struct ncclRing *ring = comms[rank]->rings+r;
      NCCLCHECK(ring->recv.transport->recv.connect(connect+2*rank+0, &ring->recv));
      NCCLCHECK(ring->send.transport->send.connect(connect+2*rank+1, &ring->send));
    }
  }
  free(rings);
  return ncclSuccess;
}

static ncclResult_t initTransportsRank(struct ncclComm* comm, ncclUniqueId* commId) {
  int rank = comm->rank;
  int nranks = comm->nRanks;
  void* commState;
  struct ncclBootstrap* bootstrap;
  NCCLCHECK(bootstrapInit(commId, rank, nranks, &bootstrap, &commState));
  
  struct ncclInfo* allInfo = (struct ncclInfo*)malloc(sizeof(struct ncclInfo)*nranks);
  fillInfo(allInfo+rank, rank);
  NCCLCHECK(bootstrap->allGather(commState, allInfo, sizeof(struct ncclInfo)));

  int *rings;
  int nrings = getRings(&rings, nranks);
  comm->nRings = nrings;

  for (int r=0; r<nrings; r++) {
    int* ringRanks = rings+r*nranks;
    struct ncclRing *ring = comm->rings+r;
    struct ncclConnect connect[2];
    NCCLCHECK(setupRing(ring, r, rank, nranks, ringRanks, allInfo, connect));
    NCCLCHECK(bootstrap->ringExchange(commState, connect, ring->userRanks[nranks-1], ring->userRanks[1], sizeof(struct ncclConnect)));
    NCCLCHECK(ring->recv.transport->recv.connect(connect+0, &ring->recv));
    NCCLCHECK(ring->send.transport->send.connect(connect+1, &ring->send));
  }
  free(rings);
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommInitRank, ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank);
ncclResult_t ncclCommInitRank(ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank) {
  if (myrank == 0) showVersion();

  initDebug();
  ncclResult_t res;

  res = wrapNvmlSymbols();
  if (res != ncclSuccess) {
    WARN("NCCL failed to initialize NVML");
    return res;
  }

  res = wrapNvmlInit();
  if (res != ncclSuccess) {
    WARN("rank %d failed to initialize nvml", myrank);
    return res;
  }

  res = commAlloc(newcomm, ndev, myrank);
  if (res != ncclSuccess) {
    WARN("rank %d failed to allocate communicator", myrank);
    return res;
  }

  res = initTransportsRank(*newcomm, &commId);
  if (res != ncclSuccess) {
    WARN("rank %d failed to init transports", myrank);
    return res;
  }

  res = devCommSetup(*newcomm);
  if (res != ncclSuccess) {
    WARN("rank %d failed to copy dcomm", myrank);
    return res;
  }

  if (wrapNvmlShutdown() != ncclSuccess)
    INFO("rank %d did not shutdown nvml properly", myrank);
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommInitAll, ncclComm_t* comms, int ndev, const int* devlist);
ncclResult_t ncclCommInitAll(ncclComm_t* comms, int ndev, const int* devlist) {
  initDebug();

  showVersion();

  ncclResult_t res;
  int savedDevice;
  int rank, cudaDev;
  ncclComm_t comm = NULL;
  char busId[13];
  nvmlDevice_t nvmlHandle;
  int affinity_set = 0;

  res = wrapNvmlSymbols();
  if (res != ncclSuccess) {
    WARN("NCCL failed to initialize NVML");
    return res;
  }

  hipGetDevice(&savedDevice);

  res = wrapNvmlInit();
  if (res != ncclSuccess) {
    WARN("nccl failed to initialize nvml");
    return res;
  }

  for(rank=0; rank<ndev; ++rank)
    comms[rank] = NULL;

  for (rank=0; rank<ndev; ++rank) {
    cudaDev = (devlist == NULL) ? rank : devlist[rank];
    if (hipSetDevice(cudaDev) != hipSuccess) {
      WARN("rank %d failed to set cuda device %d", rank, cudaDev);
      res = ncclInvalidDeviceIndex;
      goto cleanup;
    }

    // Set CPU affinity
    affinity_set = 0;
    if (hipDeviceGetPCIBusId(busId, 13, cudaDev) != hipSuccess) {
      INFO("rank %d failed to get PCI Bus Id for device %d", rank, cudaDev);
      goto skipaffinity;
    }
    if (wrapNvmlDeviceGetHandleByPciBusId(busId, &nvmlHandle) != ncclSuccess) {
      INFO("rank %d failed to get nvml handle for device %s", rank, busId);
      goto skipaffinity;
    }
    if (wrapNvmlDeviceSetCpuAffinity(nvmlHandle) != ncclSuccess) {
      INFO("rank %d failed to set affinity", rank);
      goto skipaffinity;
    }
    affinity_set = 1;
    skipaffinity:

    res = commAlloc(&comm, ndev, rank);
    if (res != ncclSuccess) {
      WARN("rank %d failed to allocate communicator", rank);
      goto cleanup;
    }
    comms[rank] = comm;

    if (affinity_set && wrapNvmlDeviceClearCpuAffinity(nvmlHandle) != ncclSuccess) {
      INFO("rank %d set but failed to clear cpu affinity", rank);
    }
  }

  res = initTransportsAll(comms, devlist, ndev);
  if (res != ncclSuccess) {
    WARN("failed to init transports");
    return res;
  }

  for(rank=0; rank<ndev; ++rank) {
    cudaDev = (devlist == NULL) ? rank : devlist[rank];
    if (hipSetDevice(cudaDev) != hipSuccess) {
      WARN("rank %d failed to set cuda device %d", rank, cudaDev);
      res = ncclInvalidDeviceIndex;
      goto cleanup;
    }
    res = devCommSetup(comms[rank]);
    if (res != ncclSuccess) {
      WARN("rank %d failed to copy dcomm", rank);
    }
  }

  res = ncclSuccess;
  goto final;

  cleanup:
  for(rank=0; rank<ndev; ++rank) {
    if(comms[rank] != NULL) {
      commFree(comms[rank]);
    }
  }

  final:
  if(wrapNvmlShutdown() != ncclSuccess)
    INFO("NCCL did not shutdown nvml properly");
  hipSetDevice(savedDevice);
  return res;
}

NCCL_API(ncclResult_t, ncclCommDestroy, ncclComm_t comm);
ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == NULL)
    return ncclSuccess;

  int savedDevice;
  CUDACHECK(hipGetDevice(&savedDevice));
  int commDevice = comm->cudaDev;

  if (savedDevice != commDevice) {
    CUDACHECK(hipSetDevice(commDevice));
  }

  commFree(comm);

  if (savedDevice != commDevice)
    CUDACHECK(hipSetDevice(savedDevice));

  return ncclSuccess;
}

NCCL_API(const char*, ncclGetErrorString, ncclResult_t code);
const char* ncclGetErrorString(ncclResult_t code) {
  switch (code) {
  case ncclSuccess                : return "no error";
  case ncclUnhandledCudaError     : return "unhandled cuda error";
  case ncclSystemError            : return "system error";
  case ncclInternalError          : return "internal error";
  case ncclInvalidDevicePointer   : return "invalid device pointer";
  case ncclInvalidRank            : return "invalid rank";
  case ncclUnsupportedDeviceCount : return "unsupported device count";
  case ncclDeviceNotFound         : return "device not found";
  case ncclInvalidDeviceIndex     : return "invalid device index";
  case ncclLibWrapperNotSet       : return "lib wrapper not initialized";
  case ncclCudaMallocFailed       : return "cuda malloc failed";
  case ncclRankMismatch           : return "parameter mismatch between ranks";
  case ncclInvalidArgument        : return "invalid argument";
  case ncclInvalidType            : return "invalid data type";
  case ncclInvalidOperation       : return "invalid reduction operations";
  }
  return "unknown result code";
}

NCCL_API(ncclResult_t, ncclCommCount, const ncclComm_t comm, int* count);
ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  *count = comm->nRanks;
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommCuDevice, const ncclComm_t comm, int* devid);
ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* devid) {
  *devid = comm->cudaDev;
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommUserRank, const ncclComm_t comm, int* rank);
ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  *rank = comm->rank;
  return ncclSuccess;
}

