#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "nvmlwrap.h"
#include "rings.h"
#include "bootstrap.h"
#include "transport.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sched.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <errno.h>

DebugLevel ncclDebugLevel;
pthread_mutex_t ncclDebugOutputLock;

int ncclPrintCRCs;

#define MAX_ASYNC_THREADS 128
thread_local pthread_t ncclThreads[MAX_ASYNC_THREADS];
thread_local int ncclThreadIndex = 0;
thread_local bool ncclThreadMode = 0;

NCCL_API(ncclResult_t, ncclGroupStart);
ncclResult_t ncclGroupStart() {
  ncclThreadMode = 1;
  return ncclSuccess;
}

struct ncclInitArgs {
  int cudaDev;
  ncclResult_t ret;
  ncclComm_t* newcomm;
  int ndev;
  ncclUniqueId commId;
  int myrank; 
};

NCCL_API(ncclResult_t, ncclGroupEnd);
ncclResult_t ncclGroupEnd() {
  int done = ncclThreadIndex;
  int doneArray[ncclThreadIndex];
  for (int i=0; i<ncclThreadIndex; i++) doneArray[i] = 0;
  while (done) {
    for (int i=0; i<ncclThreadIndex; i++) {
      struct ncclInitArgs* args;
      if (doneArray[i] == 1) continue;
      int err = pthread_tryjoin_np(ncclThreads[i], (void**)&args);
      if (err == EBUSY) continue;
      if (err != 0) return ncclSystemError;
      if (args->ret != ncclSuccess) return args->ret;
      doneArray[i] = 1;
      done--;
      free(args);
    }
  }
  ncclThreadIndex = 0;
  ncclThreadMode = 0;
  return ncclSuccess;
}

ncclResult_t ncclCommInitRankSync(ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank);

void* ncclCommInitRankThread(void* args_) {
  struct ncclInitArgs* args = (struct ncclInitArgs*)args_;
  hipSetDevice(args->cudaDev);
  args->ret = ncclCommInitRankSync(args->newcomm, args->ndev, args->commId, args->myrank);
  return args;
}

static ncclResult_t ncclCommInitRankAsync(ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank) {
  struct ncclInitArgs* args = (struct ncclInitArgs*)malloc(sizeof(struct ncclInitArgs));
  
  CUDACHECK(hipGetDevice(&args->cudaDev));
  args->newcomm = newcomm;
  args->ndev = ndev;
  memcpy(&args->commId, &commId, sizeof(commId));
  args->myrank = myrank;
  
  SYSCHECK(pthread_create(ncclThreads+ncclThreadIndex, NULL, ncclCommInitRankThread, args), "pthread_create");
  ncclThreadIndex++;
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclGetUniqueId, ncclUniqueId* out);
ncclResult_t ncclGetUniqueId(ncclUniqueId* out) {
  bootstrapGetUniqueId(out);
  if (out == NULL) {
    WARN("Error : no bootstrap available");
    return ncclInternalError;
  }
  return ncclSuccess;
}

static ncclResult_t commFree(ncclComm_t comm) {
  if (comm == NULL)
    return ncclSuccess;

  for (int ring=0; ring<comm->nRings; ring++) {
    free(comm->rings[ring].userRanks);
    CUDACHECK(hipFree(comm->rings[ring].devUserRanks));
    NCCLCHECK(comm->rings[ring].send.transport->send.free(comm->rings[ring].send.transportResources));
    NCCLCHECK(transportDestroyProxy(&comm->rings[ring].send));
    NCCLCHECK(comm->rings[ring].recv.transport->recv.free(comm->rings[ring].recv.transportResources));
    NCCLCHECK(transportDestroyProxy(&comm->rings[ring].recv));
  }

  if (comm->doneEvent != NULL)
    CUDACHECK(hipEventDestroy(comm->doneEvent));

  free(comm);
  return ncclSuccess;
}

static ncclResult_t commAlloc(ncclComm_t* comret, int ndev, int rank) {
  if (ndev < 1) {
    WARN("invalid device count (%d) requested", ndev);
    return ncclUnsupportedDeviceCount;
  }
  if (rank >= ndev || rank < 0) {
    WARN("rank %d exceeds ndev=%d", rank, ndev);
    return ncclInvalidRank;
  }

  // Try to create a CUDA object right away. If there is something wrong with
  // the device we're on (failure cause #1) , better know it early.
  hipEvent_t doneEvent;
  CUDACHECK(hipEventCreateWithFlags(&doneEvent, hipEventDisableTiming));

  struct ncclComm* comm = (struct ncclComm*)malloc(sizeof(struct ncclComm));
  if (comm == NULL) {
    WARN("comm allocation failed");
    return ncclSystemError;
  }
  memset(comm, 0, sizeof(struct ncclComm));

  comm->rank = rank;
  comm->nRanks = ndev;
  hipGetDevice(&comm->cudaDev);
  comm->doneEvent = doneEvent;

  *comret = comm;
  return ncclSuccess;
}

static ncclResult_t devCommSetup(ncclComm_t comm) {
  // Fully duplicate the comm on the device
  if (hipMalloc(&comm->devComm, sizeof(struct ncclComm)) != hipSuccess) {
    WARN("failed to allocated device comm");
    return ncclCudaMallocFailed;
  }
  // Copy the comm on the device
  if (hipMemcpy(comm->devComm, comm, sizeof(struct ncclComm), hipMemcpyHostToDevice) != hipSuccess) {
    WARN("failed to copy device comm");
    return ncclUnhandledCudaError;
  }
  // Copy userRanks
  for (int r=0; r<comm->nRings; r++) {
    CUDACHECK(hipMemcpy(comm->rings[r].devUserRanks, comm->rings[r].userRanks, comm->nRanks*sizeof(int), hipMemcpyHostToDevice));
  }
  return ncclSuccess;
}

static void showVersion() {
  static int shown = 0;
  if (shown == 0 && ncclDebugLevel >= VERSION) {
    printf("NCCL version %d.%d.%d compiled with CUDA %d.%d\n", NCCL_MAJOR, NCCL_MINOR, NCCL_PATCH, CUDA_MAJOR, CUDA_MINOR);
    fflush(stdout);
    shown = 1;
  }
}

static ncclResult_t fillInfo(struct ncclInfo* info, int rank) {
  for (int t=0; t<NTRANSPORTS; t++) {
    NCCLCHECK(ncclTransports[t].fillInfo(info->tinfo+t, rank));
  }
  return ncclSuccess;
}

template <int type>
static ncclResult_t selectTransport(struct ncclInfo* myInfo, struct ncclInfo* peerInfo, struct ncclConnect* connect, struct ncclTransport** transportRet, struct ncclRing* ring) {
  for (int t=0; t<NTRANSPORTS; t++) {
    struct ncclTransport *transport = ncclTransports+t;
    struct ncclTransportComm* transportComm = type == 1 ? &transport->send : &transport->recv;
    int ret = 0;
    NCCLCHECK(transport->canConnect(&ret, myInfo->tinfo+t, peerInfo->tinfo+t));
    if (ret > 0) {
      NCCLCHECK(transportComm->setup(myInfo->tinfo+t, peerInfo->tinfo+t, connect, ring));
      *transportRet = transport;
      return ncclSuccess;
    }
  }
  WARN("No transport found !");
  *transportRet = NULL;
  return ncclInternalError;
}

static ncclResult_t setupSendRecv(struct ncclRing* ring) {
  const char* str = getenv("NCCL_BUFFSIZE");
  int buffSize;
  if (str != NULL) {
    errno = 0;
    buffSize = strtol(str, NULL, 10);
    if (errno == ERANGE || buffSize == 0) {
      INFO("invalid NCCL_BUFFSIZE: %s, using default %lu",
          str, DEFAULT_BUFFER_SIZE_BYTES);
      buffSize = DEFAULT_BUFFER_SIZE_BYTES;
    }
  } else {
    buffSize = DEFAULT_BUFFER_SIZE_BYTES;
  }
  ring->buffSize = buffSize;
  const int size = ring->devMemSize = offsetof(struct ncclSendRecvMem, buff)+buffSize;
  struct ncclSendRecvMem* mem;
  CUDACHECK(hipMalloc(&mem, size));
  CUDACHECK(hipMemset(mem, 0, size));
  ring->devMem = mem;
  ring->recv.conn.buff = (char*)&mem->buff;
  ring->recv.conn.tail = &mem->tail;
  ring->recv.conn.opCount = &mem->opCount;
  ring->recv.conn.direct = 0;
  ring->send.conn.head = &mem->head;
  ring->send.conn.direct = 0;
  return ncclSuccess;
}

static ncclResult_t fillConnect(struct ncclInfo* allInfo, int nranks, int rank, int* connectTransport, int* connectValue) {
  for (int r=0; r<nranks; r++) {
    connectTransport[r] = -1;
    for (int t=0; t<NTRANSPORTS; t++) {
      NCCLCHECK(ncclTransports[t].canConnect(connectValue+r, allInfo[rank].tinfo+t, allInfo[r].tinfo+t));
      if (connectValue[r] > 0) {
        connectTransport[r] = t;
        break;
      }
    }
  }
  return ncclSuccess;
}

static ncclResult_t setupRing(struct ncclComm* comm, struct ncclRing* ring, int ringid, int rank, int nranks, int* ringRanks, struct ncclInfo* allInfo, struct ncclConnect* connect) { 
  ring->id = ringid;
  // Reorganize ranks to start with rank.
  int shift;
  for (shift = 0; shift<nranks; shift++) {
    if (ringRanks[shift] == rank) {
      ring->rank = shift;
      break;
    }
  }
  CUDACHECK(hipMalloc(&ring->devUserRanks, nranks*sizeof(int)));
  ring->userRanks = (int*)malloc(nranks*sizeof(int));
  for (int i=0; i<nranks; i++) {
    ring->userRanks[i] = ringRanks[(i+shift)%nranks];
  }
  int prev = ring->userRanks[nranks-1];
  int next = ring->userRanks[1];

  setupSendRecv(ring);
  NCCLCHECK(selectTransport<0>(allInfo+rank, allInfo+prev, connect+0, &ring->recv.transport, ring));
  NCCLCHECK(selectTransport<1>(allInfo+rank, allInfo+next, connect+1, &ring->send.transport, ring));
  NCCLCHECK(transportCreateProxy(0, ring, comm));
  NCCLCHECK(transportCreateProxy(1, ring, comm));
  return ncclSuccess;
}

static void swap(void* mem1, void* mem2, int size) {
  char tmp[size];
  memcpy(tmp, mem1, size); memcpy(mem1, mem2, size); memcpy(mem2, tmp, size);
}

#define MAXWIDTH 20
#define PREFIXLEN 15
#define STRLENGTH (PREFIXLEN+4*MAXWIDTH)
void dumpMatrix(int* connectMatrix, int nranks) {
  char line[STRLENGTH+1];
  line[STRLENGTH] = '\0';
  memset(line, ' ', STRLENGTH);
  for (int j=0; j<nranks && j<MAXWIDTH; j++) sprintf(4+line+4*j, " %3d", j);
  INFO(line);
  for (int i=0; i<nranks; i++) {
    memset(line, ' ', STRLENGTH);
    sprintf(line, "%3d ", i);
    for (int j=0; j<nranks && j<MAXWIDTH; j++) sprintf(4+line+4*j, " %3d", connectMatrix[i*nranks+j]);
    INFO(line);
  }
}

void dumpLine(int* values, int nranks, const char* prefix) {
  int prefixlen = strlen(prefix);
  char line[STRLENGTH+1];
  line[STRLENGTH] = '\0';
  memset(line, ' ', STRLENGTH);
  strncpy(line, prefix, PREFIXLEN);
  for (int i=0; i<nranks && i<MAXWIDTH; i++) sprintf(line+prefixlen+4*i, " %3d", values[i]);
  INFO(line);
}

static ncclResult_t buildRings(int nrings, int* rings, int rank, int nranks, int* prev, int* next) {
  for (int r=0; r<nrings; r++) {
    char prefix[30];
    /*sprintf(prefix, "[%d] Ring %d Prev : ", rank, r);
    dumpLine(prev+r*nranks, nranks, prefix);
    sprintf(prefix, "[%d] Ring %d Next : ", rank, r);
    dumpLine(next+r*nranks, nranks, prefix);*/

    int current = rank;
    for (int i=0; i<nranks; i++) {
      rings[r*nranks+i] = current;
      current = next[r*nranks+current];
    }
    sprintf(prefix, "[%d] Ring %d : ", rank, r);
    dumpLine(rings+r*nranks, nranks, prefix);
    if (current != rank) {
      WARN("Error : ring %d does not loop back to start (%d != %d)", r, current, rank);
      return ncclInternalError;
    }
    // Check that all ranks are there
    for (int i=0; i<nranks; i++) {
      int found = 0;
      for (int j=0; j<nranks; j++) {
        if (rings[r*nranks+j] == i) {
          found = 1;
          break;
        }
      }
      if (found == 0) {
        WARN("Error : ring %d does not contain rank %d", r, i);
        return ncclInternalError;
      }
    }
  }
  return ncclSuccess;
}

static ncclResult_t initTransportsRank(struct ncclComm* comm, ncclUniqueId* commId) {
  int rank = comm->rank;
  int nranks = comm->nRanks;
  void* commState;
  struct ncclBootstrap* bootstrap;
  NCCLCHECK(bootstrapInit(commId, rank, nranks, &bootstrap, &commState));
  
  struct ncclInfo* allInfo = (struct ncclInfo*)malloc(sizeof(struct ncclInfo)*nranks);
  fillInfo(allInfo+rank, rank);
  NCCLCHECK(bootstrap->allGather(commState, allInfo, sizeof(struct ncclInfo)));
  int connectTransport[nranks*nranks];
  int connectValue[nranks*nranks];
  NCCLCHECK(fillConnect(allInfo, nranks, rank, connectTransport+nranks*rank, connectValue+nranks*rank));
  NCCLCHECK(bootstrap->allGather(commState, connectTransport, nranks*(sizeof(int))));
  NCCLCHECK(bootstrap->allGather(commState, connectValue, nranks*(sizeof(int))));
  //if (rank == 0) dumpMatrix(connectTransport, nranks);
  //if (rank == 0) dumpMatrix(connectValue, nranks);

  // Get my rings
  int nrings;
  int prev[nranks*MAXRINGS];
  int next[nranks*MAXRINGS];
  NCCLCHECK(ncclGetRings(&nrings, rank, nranks, connectTransport, connectValue, prev, next));

  // Find min nrings across ranks
  int allNrings[nranks];
  allNrings[rank] = nrings;
  NCCLCHECK(bootstrap->allGather(commState, allNrings, sizeof(int)));
  for (int i=0; i<nranks; i++)
    nrings = min(allNrings[i], nrings);

  // Exchange data with others to build complete rings
  comm->nRings = nrings;
  for (int r=0; r<nrings; r++) {
    NCCLCHECK(bootstrap->allGather(commState, prev+r*nranks, sizeof(int)));
    NCCLCHECK(bootstrap->allGather(commState, next+r*nranks, sizeof(int)));
  }
  int rings[nranks*MAXRINGS];
  NCCLCHECK(buildRings(nrings, rings, rank, nranks, prev, next));

  // Connect with prev/next for each ring
  for (int r=0; r<nrings; r++) {
    int* ringRanks = rings+r*nranks;
    struct ncclRing *ring = comm->rings+r;
    struct ncclConnect connect[2];
    NCCLCHECK(setupRing(comm, ring, r, rank, nranks, ringRanks, allInfo, connect));
    NCCLCHECK(bootstrap->ringExchange(commState, connect, ring->userRanks[nranks-1], ring->userRanks[1], sizeof(struct ncclConnect)));
    NCCLCHECK(ring->recv.transport->recv.connect(connect+0, &ring->recv));
    NCCLCHECK(ring->send.transport->send.connect(connect+1, &ring->send));
  }
  free(allInfo);
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommInitRank, ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank);
ncclResult_t ncclCommInitRank(ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank) {
  return ncclThreadMode ? 
    ncclCommInitRankAsync(newcomm, ndev, commId, myrank) :
    ncclCommInitRankSync(newcomm, ndev, commId, myrank);
}

ncclResult_t ncclCommInitRankSync(ncclComm_t* newcomm, int ndev, ncclUniqueId commId, int myrank) {
  if (myrank == 0) showVersion();

  initDebug();
  ncclResult_t res;

  res = wrapNvmlSymbols();
  if (res != ncclSuccess) {
    WARN("NCCL failed to initialize NVML");
    return res;
  }

  res = wrapNvmlInit();
  if (res != ncclSuccess) {
    WARN("rank %d failed to initialize nvml", myrank);
    return res;
  }

  res = commAlloc(newcomm, ndev, myrank);
  if (res != ncclSuccess) {
    WARN("rank %d failed to allocate communicator", myrank);
    return res;
  }

  res = initTransportsRank(*newcomm, &commId);
  if (res != ncclSuccess) {
    WARN("rank %d failed to init transports", myrank);
    return res;
  }

  res = devCommSetup(*newcomm);
  if (res != ncclSuccess) {
    WARN("rank %d failed to copy dcomm", myrank);
    return res;
  }

  if (wrapNvmlShutdown() != ncclSuccess)
    INFO("rank %d did not shutdown nvml properly", myrank);
  return ncclSuccess;
}

static ncclResult_t initTransportsAll(struct ncclComm** comms, const int* devs, int nranks) {
  struct ncclInfo* allInfo = (struct ncclInfo*)malloc(sizeof(struct ncclInfo)*nranks);
  for (int rank=0; rank<nranks; rank++) {
    hipSetDevice(devs[rank]);
    fillInfo(allInfo+rank, rank);
  }

  int connectTransport[nranks*nranks];
  int connectValue[nranks*nranks];
  for (int rank=0; rank<nranks; rank++)
    NCCLCHECK(fillConnect(allInfo, nranks, rank, connectTransport+nranks*rank, connectValue+nranks*rank));
  
  int nrings;
  int nringsFinal = MAXRINGS;
  int prev[nranks*MAXRINGS];
  int prevFinal[nranks*MAXRINGS];
  int next[nranks*MAXRINGS];
  int nextFinal[nranks*MAXRINGS];
  for (int rank=0; rank<nranks; rank++) {
    NCCLCHECK(ncclGetRings(&nrings, rank, nranks, connectTransport, connectValue, prev, next));
    nringsFinal = min(nrings, nringsFinal);
    for (int ring=0; ring<nrings; ring++) {
      int index = ring*nranks+rank;
      prevFinal[index] = prev[index];
      nextFinal[index] = next[index];
    }
  }
  nrings = nringsFinal;
  int rings[nranks*MAXRINGS];
  NCCLCHECK(buildRings(nrings, rings, 0, nranks, prevFinal, nextFinal));

  for (int rank=0; rank<nranks; rank++)
    comms[rank]->nRings = nrings;

  for (int r=0; r<nrings; r++) {
    struct ncclConnect connect[2*nranks];
    int* ringRanks = rings+r*nranks;
    for (int rank=0; rank<nranks; rank++) {
      CUDACHECK(hipSetDevice(devs[rank]));
      struct ncclRing *ring = comms[rank]->rings+r;
      NCCLCHECK(setupRing(comms[rank], ring, r, rank, nranks, ringRanks, allInfo, connect+2*rank));
    }
    // RingExchange connect information
    for (int rank=0; rank<nranks; rank++) {
      // Swap rank->prev and prevRank->next
      struct ncclRing *ring = comms[rank]->rings+r;
      int prevRank = ring->userRanks[nranks-1];
      struct ncclConnect* prevRankNextConnect = connect+2*prevRank+1;
      struct ncclConnect* rankPrevConnect = connect+2*rank;
      swap(prevRankNextConnect, rankPrevConnect, sizeof(struct ncclConnect));
    }
    for (int rank=0; rank<nranks; rank++) {
      CUDACHECK(hipSetDevice(devs[rank]));
      struct ncclRing *ring = comms[rank]->rings+r;
      NCCLCHECK(ring->recv.transport->recv.connect(connect+2*rank+0, &ring->recv));
      NCCLCHECK(ring->send.transport->send.connect(connect+2*rank+1, &ring->send));
    }
  }
  free(allInfo);
  return ncclSuccess;
}


NCCL_API(ncclResult_t, ncclCommInitAll, ncclComm_t* comms, int ndev, const int* devlist);
ncclResult_t ncclCommInitAll(ncclComm_t* comms, int ndev, const int* devlist) {
  initDebug();

  showVersion();

  ncclResult_t res;
  int savedDevice;
  int rank, cudaDev;
  ncclComm_t comm = NULL;
  char busId[13];
  nvmlDevice_t nvmlHandle;
  int affinity_set = 0;
  int ncclDevList[ndev];
  for (int i=0; i<ndev; i++) {
    ncclDevList[i] = devlist ? devlist[i] : i;
  }

  res = wrapNvmlSymbols();
  if (res != ncclSuccess) {
    WARN("NCCL failed to initialize NVML");
    return res;
  }

  hipGetDevice(&savedDevice);

  res = wrapNvmlInit();
  if (res != ncclSuccess) {
    WARN("nccl failed to initialize nvml");
    return res;
  }

  for(rank=0; rank<ndev; ++rank)
    comms[rank] = NULL;

  for (rank=0; rank<ndev; ++rank) {
    cudaDev = ncclDevList[rank];
    if (hipSetDevice(cudaDev) != hipSuccess) {
      WARN("rank %d failed to set cuda device %d", rank, cudaDev);
      res = ncclInvalidDeviceIndex;
      goto cleanup;
    }

    // Set CPU affinity
    affinity_set = 0;
    if (hipDeviceGetPCIBusId(busId, 13, cudaDev) != hipSuccess) {
      INFO("rank %d failed to get PCI Bus Id for device %d", rank, cudaDev);
      goto skipaffinity;
    }
    if (wrapNvmlDeviceGetHandleByPciBusId(busId, &nvmlHandle) != ncclSuccess) {
      INFO("rank %d failed to get nvml handle for device %s", rank, busId);
      goto skipaffinity;
    }
    if (wrapNvmlDeviceSetCpuAffinity(nvmlHandle) != ncclSuccess) {
      INFO("rank %d failed to set affinity", rank);
      goto skipaffinity;
    }
    affinity_set = 1;
    skipaffinity:

    res = commAlloc(&comm, ndev, rank);
    if (res != ncclSuccess) {
      WARN("rank %d failed to allocate communicator", rank);
      goto cleanup;
    }
    comms[rank] = comm;

    if (affinity_set && wrapNvmlDeviceClearCpuAffinity(nvmlHandle) != ncclSuccess) {
      INFO("rank %d set but failed to clear cpu affinity", rank);
    }
  }

  res = initTransportsAll(comms, ncclDevList, ndev);
  if (res != ncclSuccess) {
    WARN("failed to init transports");
    return res;
  }

  for(rank=0; rank<ndev; ++rank) {
    cudaDev = ncclDevList[rank];
    if (hipSetDevice(cudaDev) != hipSuccess) {
      WARN("rank %d failed to set cuda device %d", rank, cudaDev);
      res = ncclInvalidDeviceIndex;
      goto cleanup;
    }
    res = devCommSetup(comms[rank]);
    if (res != ncclSuccess) {
      WARN("rank %d failed to copy dcomm", rank);
    }
  }

  res = ncclSuccess;
  goto final;

  cleanup:
  for(rank=0; rank<ndev; ++rank) {
    if(comms[rank] != NULL) {
      commFree(comms[rank]);
    }
  }

  final:
  if(wrapNvmlShutdown() != ncclSuccess)
    INFO("NCCL did not shutdown nvml properly");
  hipSetDevice(savedDevice);
  return res;
}

NCCL_API(ncclResult_t, ncclCommDestroy, ncclComm_t comm);
ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == NULL)
    return ncclSuccess;

  int savedDevice;
  CUDACHECK(hipGetDevice(&savedDevice));
  int commDevice = comm->cudaDev;

  if (savedDevice != commDevice) {
    CUDACHECK(hipSetDevice(commDevice));
  }

  commFree(comm);

  if (savedDevice != commDevice)
    CUDACHECK(hipSetDevice(savedDevice));

  return ncclSuccess;
}

NCCL_API(const char*, ncclGetErrorString, ncclResult_t code);
const char* ncclGetErrorString(ncclResult_t code) {
  switch (code) {
  case ncclSuccess                : return "no error";
  case ncclUnhandledCudaError     : return "unhandled cuda error";
  case ncclSystemError            : return "system error";
  case ncclInternalError          : return "internal error";
  case ncclInvalidDevicePointer   : return "invalid device pointer";
  case ncclInvalidRank            : return "invalid rank";
  case ncclUnsupportedDeviceCount : return "unsupported device count";
  case ncclDeviceNotFound         : return "device not found";
  case ncclInvalidDeviceIndex     : return "invalid device index";
  case ncclLibWrapperNotSet       : return "lib wrapper not initialized";
  case ncclCudaMallocFailed       : return "cuda malloc failed";
  case ncclRankMismatch           : return "parameter mismatch between ranks";
  case ncclInvalidArgument        : return "invalid argument";
  case ncclInvalidType            : return "invalid data type";
  case ncclInvalidOperation       : return "invalid reduction operations";
  }
  return "unknown result code";
}

NCCL_API(ncclResult_t, ncclCommCount, const ncclComm_t comm, int* count);
ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  *count = comm->nRanks;
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommCuDevice, const ncclComm_t comm, int* devid);
ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* devid) {
  *devid = comm->cudaDev;
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclCommUserRank, const ncclComm_t comm, int* rank);
ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  *rank = comm->rank;
  return ncclSuccess;
}

