/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "transport.h"
#include "nvmlwrap.h"
#include "net.h"
#include "gdcopy.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define NET_MAX_IFS 8
struct netInfo {
  int rank;
  int ndev;
  int scores[NET_MAX_IFS];
};

struct netConnectInfo {
  ncclNetHandle_t netHandle;
};

struct netSendResources {
  void* netSendComm;
  struct ncclSendRecvMem* hostMem;
  struct ncclSendRecvMem* devHostMem;
  struct ncclSendRecvMem* hostDevMem;
  int netDev;
  bool cudaSupport;
  struct ncclSendRecvMem* devNetMem;
};

struct netRecvResources {
  void* netListenComm;
  void* netRecvComm;
  struct ncclSendRecvMem* hostMem;
  struct ncclSendRecvMem* devHostMem;
  struct ncclSendRecvMem* hostDevMem;
  int netDev;
  bool cudaSupport;
};

/* Fill information necessary to exchange between ranks to choose whether or not
 * to use this transport */
ncclResult_t netFillInfo(ncclTinfo_t* opaqueInfo, int rank) {
  struct netInfo* info = (struct netInfo*)opaqueInfo;
  static_assert(sizeof(struct netInfo) <= sizeof(ncclTinfo_t), "NET Info too large");
  info->rank = rank;
  int *distances;
  NCCLCHECK(ncclNetDevices(&info->ndev, &distances));
  if (info->ndev == 0) {
    WARN("Error : Network returned 0 device");
    return ncclSystemError;
  }
  if (info->ndev > NET_MAX_IFS) info->ndev = NET_MAX_IFS;
  for (int d=0; d<info->ndev; d++) info->scores[d] = distances[d];
  free(distances);
  return ncclSuccess;
}

/* Determine if we can communicate with the peer */
ncclResult_t netCanConnect(int* ret, ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo) {
  ret[0] = 0;
  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  for (int d=0; d<myInfo->ndev; d++) {
    // Keep 2 bits of distance
    ret[0] |= ((myInfo->scores[d]&0x7)<<(3*d));
  }
  return ncclSuccess;
}

static inline void groupLastFirst(int nranks, int* groups, int group1, int group2, int* values, int ring, int* rank1, int* rank2, int minScore) {
  // Find last of group1
  for (int r1 = nranks-1; r1>=0; r1--) {
    if (groups[r1] == group1) {
      // Find first of group2
      for (int r2 = 0; r2<nranks; r2++) {
        if (groups[r2] == group2) {
          // Check both can talk through that device = ring
          if ((values[r1*nranks+r2] & (0x7<<(3*ring))) >= minScore &&
              (values[r2*nranks+r1] & (0x7<<(3*ring))) >= minScore) {
            *rank1 = r1;
            *rank2 = r2;
            return;
          }
        }
      }
    }
  }
  *rank1 = -1;
  *rank2 = -1;
}

ncclResult_t netGetRings(int nranks, int ngroups, int* groups, int* values, int* nringsRet, int* prev, int* next, int minScore) {
  for (int ring = 0; ring<*nringsRet; ring++) {
    for (int group = 0; group<ngroups; group++) {
      // Check if this group is already connected
      int skip = 0;
      for (int rank = 0; rank<nranks; rank++) {
        if (groups[rank] == group && next[ring*nranks+rank] != -1) skip = 1;
      }
      if (skip) continue;

      int source = -1, destination = -1;
      if (ring % 2 == 0) {
        int nextGroup = (group+1)%ngroups;
        groupLastFirst(nranks, groups, group, nextGroup, values, ring, &source, &destination, minScore);
      } else {
        int prevGroup = (group-1+ngroups)%ngroups;
        groupLastFirst(nranks, groups, prevGroup, group, values, ring, &destination, &source, minScore);
      }
      if (source == -1 || destination == -1) {
        *nringsRet = ring;
        return ncclSuccess;
      }
      next[ring*nranks+source] = destination;
      prev[ring*nranks+destination] = source;
    }
  }
  return ncclSuccess;
}

static ncclResult_t netHostAlloc(struct ncclSendRecvMem** ptr, size_t size) {
  // Allocate memory close to the device we are using
  CUDACHECK(hipHostAlloc(ptr, size, hipHostMallocMapped));
  return ncclSuccess;
}

int getDev(int ringId, int nDev, int* scores) {
  int maxScore = 0;
  for (int d=0; d<nDev; d++) if (scores[d] > maxScore) maxScore = scores[d];
  int skip = ringId+1;
  while (skip) {
    for (int d=0; d<nDev; d++) {
      if (scores[d] == maxScore) {
        skip--;
        if (skip == 0) return d;
      }
    }
  }
  return 0;
}

/* Determine if we will use this transport for this peer and return connect
 * information for this peer */
ncclResult_t netSendSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netSendResources* resources = (struct netSendResources*) malloc(sizeof(struct netSendResources));
  ring->send.transportResources = resources;
  resources->hostDevMem = NULL; //(struct ncclSendRecvMem*)gdptr(ring->devMem, ring->buffSize);

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->scores);
  int flags;
  NCCLCHECK(ncclNetPtrSupport(resources->netDev, &flags));
  static int useGDRforReads = -1;
  if (useGDRforReads == -1) {
    char* str = getenv("NCCL_NET_GDR_READ");
    useGDRforReads = str ? atoi(str) : 0;
  }
  resources->cudaSupport = (useGDRforReads == 1) && (flags & NCCL_PTR_CUDA) ? true : false;

  int size = offsetof(struct ncclSendRecvMem, buff)+ring->buffSize;
  if (resources->cudaSupport) {
    CUDACHECK(hipMalloc(&resources->devNetMem, size));
  }
  NCCLCHECK(netHostAlloc(&resources->hostMem, size));
  CUDACHECK(hipHostGetDevicePointer(&resources->devHostMem, resources->hostMem, 0));

  return ncclSuccess;
}

ncclResult_t netRecvSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netRecvResources* resources = (struct netRecvResources*) malloc(sizeof(struct netRecvResources));
  ring->recv.transportResources = resources;
  resources->hostDevMem = NULL; //(struct ncclSendRecvMem*)gdptr(ring->devMem, ring->buffSize);

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->scores);
  int flags;
  NCCLCHECK(ncclNetPtrSupport(resources->netDev, &flags));
  resources->cudaSupport = (flags & NCCL_PTR_CUDA) ? true : false;

  int size = offsetof(struct ncclSendRecvMem, buff)+ring->buffSize;
  NCCLCHECK(netHostAlloc(&resources->hostMem, size));
  CUDACHECK(hipHostGetDevicePointer(&resources->devHostMem, resources->hostMem, 0));
  
  struct netInfo* peerInfo = (struct netInfo*)peerOpaqueInfo;
  INFO("%d -> %d via NET/%s/%d%s%s", peerInfo->rank, myInfo->rank, ncclNetName(), resources->netDev,
      resources->cudaSupport ? "/GDRDMA" : "", 
      (resources->hostDevMem != NULL) ? "/GDCopy" : "");
  struct netConnectInfo* info = (struct netConnectInfo*) connectInfo;
  NCCLCHECK(ncclNetListen(resources->netDev, &info->netHandle, &resources->netListenComm));
  return ncclSuccess;
}

ncclResult_t netSendConnect(struct ncclConnect* connectInfo, struct ncclConnector* send) {
  // Setup device pointers
  struct netSendResources* resources = (struct netSendResources*)send->transportResources;

  if (resources->cudaSupport) {
    send->conn.buff = resources->devNetMem->buff;
  } else {
    send->conn.buff = resources->devHostMem->buff;
  }
  send->conn.tail = &resources->devHostMem->tail;
  send->conn.opCount = &resources->devHostMem->opCount;
  send->conn.fifo = resources->devHostMem->sizesFifo;

  if (resources->hostDevMem == NULL)
    send->conn.head = &resources->devHostMem->head;

  // Connect to remote peer
  struct netConnectInfo* info = (struct netConnectInfo*)connectInfo;
  NCCLCHECK(ncclNetConnect(resources->netDev, info->netHandle, &resources->netSendComm));
  return ncclSuccess;
}

/* Connect to this peer */
ncclResult_t netRecvConnect(struct ncclConnect* connectInfo, struct ncclConnector* recv) {
  // Setup device pointers
  struct netRecvResources* resources = (struct netRecvResources*)recv->transportResources;

  recv->conn.head = &resources->devHostMem->head;

  if (resources->cudaSupport == false)
    recv->conn.buff = resources->devHostMem->buff;

  if (resources->hostDevMem == NULL) {
    recv->conn.tail = &resources->devHostMem->tail;
    recv->conn.opCount = &resources->devHostMem->opCount;
  }

  // Finish connection establishment
  NCCLCHECK(ncclNetAccept(resources->netListenComm, &resources->netRecvComm));
  NCCLCHECK(ncclNetCloseListen(resources->netListenComm));

  // Setup remote MPI rank / tag
  return ncclSuccess;
}

ncclResult_t netSendFree(void* transportResources) {
  struct netSendResources* resources = (struct netSendResources*)transportResources;
  CUDACHECK(hipHostFree(resources->hostMem));
  // TODO : unmap hostDevMem
  free(resources);
  return ncclSuccess;
}

ncclResult_t netRecvFree(void* transportResources) {
  struct netRecvResources* resources = (struct netRecvResources*)transportResources;
  CUDACHECK(hipHostFree(resources->hostMem));
  // TODO : unmap hostDevMem
  free(resources);
  return ncclSuccess;
}

ncclResult_t netSendProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netSendResources* resources = (struct netSendResources*) (ring->send.transportResources);
  volatile int* prevTail = &resources->hostMem->tail;
  int* prevHead = resources->hostDevMem ? &resources->hostDevMem->head : &resources->hostMem->head;
  char* localBuff = resources->cudaSupport ? resources->devNetMem->buff : resources->hostMem->buff;
  int* sizesFifo = resources->hostMem->sizesFifo;
  int buffSize = ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  // Update in case we skipped some collectives
  resources->hostMem->opCount = args->opCount;

  int head = 0;
  int tail = 0;

  int idle = 0;
  void* requests[args->substeps];
  while (tail < args->nsteps) {
    idle++;
    while (head != *prevTail) {
      // Send through network
      int slot = head%args->substeps;
      NCCLCHECK(ncclNetIsend(resources->netSendComm, localBuff+slot*sliceSize, sizesFifo[slot], NCCL_PTR_HOST, requests+slot));
      head++;
      idle = 0;
    }
    if (tail < head) {
      int done;
      int slot = tail%args->substeps;
      NCCLCHECK(ncclNetTest(requests[slot], &done, NULL));
      if (done) {
        tail++;
        *prevHead = tail;
        idle = 0;
      }
      if (idle) transportProxyIdle(idle);
    }
  }

  // Reset
  *prevTail = 0;
  resources->hostMem->opCount = args->opCount+1;
  return ncclSuccess;
}

ncclResult_t netRecvProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netRecvResources* resources = (struct netRecvResources*) (ring->recv.transportResources);

  int* nextOpCount = resources->hostDevMem ? &resources->hostDevMem->opCount : &resources->hostMem->opCount;
  transportProxyWait([=] { return *nextOpCount >= args->opCount; });

  volatile int* nextHead = &resources->hostMem->head;
  int ptrType = resources->cudaSupport ? NCCL_PTR_CUDA : NCCL_PTR_HOST;
  char* localBuff = resources->cudaSupport ? ring->devMem->buff : resources->hostMem->buff;
  char* nextBuff = (resources->cudaSupport == false && resources->hostDevMem) ? resources->hostDevMem->buff : NULL;
  int* nextTail = resources->hostDevMem ? &resources->hostDevMem->tail : &resources->hostMem->tail;

  int buffSize = ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  int head = 0;
  int tail = 0;

  int idle = 0;
  void* requests[args->substeps];
  while (*nextHead < args->nsteps) {
    idle++;
    if ((*nextHead > tail - args->substeps) && (tail < args->nsteps)) {
      int slot = tail%args->substeps;
      NCCLCHECK(ncclNetIrecv(resources->netRecvComm, localBuff+slot*sliceSize, sliceSize, ptrType, requests+slot));
      tail++;
      idle = 0;
    }
    if (tail > head) {
      int done;
      int slot = head%args->substeps;
      int size;
      NCCLCHECK(ncclNetTest(requests[slot], &done, &size));
      if (done) {
        if (nextBuff) memcpy(nextBuff+slot*sliceSize, localBuff+slot*sliceSize, size);
        head++;
        *nextTail = head;
      }
      idle = 0;
    }
    if (idle) transportProxyIdle(idle);
  }

  // Wait for last ack and reset
  transportProxyWait([=] { return *nextHead == head; });
  *nextHead = 0;

  return ncclSuccess;
}

struct ncclTransport netTransport = {
  "NET",
  netFillInfo,
  netCanConnect,
  netGetRings,
  { netSendSetup, netSendConnect, netSendFree, netSendProxy },
  { netRecvSetup, netRecvConnect, netRecvFree, netRecvProxy }
};
