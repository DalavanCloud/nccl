/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "transport.h"
#include <hip/hip_runtime.h>
#include "socket.h"
#include <assert.h>

struct socketInfo {
  int rank;
  int listen_fd;
  struct socketAddress connect_addr;
};

struct socketSendResources {
  int fd;
  hipStream_t stream;
  struct ncclSendRecvMem* hostMem;
  struct ncclSendRecvMem* devHostMem;
};

#define MAXSTEPS 8

struct socketRecvResources {
  int listen_fd;
  int fd;
  hipStream_t stream;
  hipEvent_t syncEvent[MAXSTEPS];
  struct ncclSendRecvMem* hostMem;
  struct ncclSendRecvMem* devHostMem;
};

/* Fill information necessary to exchange between ranks to choose whether or not
 * to use this transport */
ncclResult_t socketFillInfo(ncclTinfo_t* opaqueInfo, int rank) {
  struct socketInfo* info = (struct socketInfo*)opaqueInfo;
  static_assert(sizeof(struct socketInfo) <= sizeof(ncclTinfo_t), "socket Info too large");
  info->rank = rank;
  info->listen_fd = -1;
  return ncclSuccess;
}

ncclResult_t socketCreateListen(struct socketInfo* info, char* ifname) {
  if (info->listen_fd == -1) {
    NCCLCHECK(createListenSocket(&info->listen_fd, &info->connect_addr.port));
    NCCLCHECK(getIpAddr(&(info->connect_addr.ip_addr), ifname));
  }
  return ncclSuccess;
}

/* Determine if we can communicate with the peer */
ncclResult_t socketCanConnect(int* ret, ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo) {
  *ret = 1;
  return ncclSuccess;
}

ncclResult_t socketGetRings(int nranks, int ngroups, int* groups, int* values, int* nringsRet, int* prev, int* next, int pattern) {
  if (pattern >= 2) {
    *nringsRet = 0;
    return ncclSuccess;
  }
  *nringsRet = 1;
  for (int ring = 0; ring<*nringsRet; ring++) {
    for (int group = 0; group<ngroups; group++) {
      // Check if this group is already connected
      int skip = 0;
      for (int rank = 0; rank<nranks; rank++) {
        if (groups[rank] == group && next[ring*nranks+rank] != -1) skip = 1;
      }
      if (skip) continue;

      int nextGroup = (group+1)%ngroups;
      int source = groupLast(nranks, groups, group);
      int destination = groupFirst(nranks, groups, nextGroup);
      if (source == -1 || destination == -1) {
        WARN("source %d dest %d, stopping\n", source, destination);
        *nringsRet = ring;
        return ncclSuccess;
      }
      next[ring*nranks+source] = destination;
      prev[ring*nranks+destination] = source;
    }
  }
  return ncclSuccess;
}

/* Create and return connect structures for this peer to connect to me */
ncclResult_t socketSendSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct socketSendResources* resources = (struct socketSendResources*) malloc(sizeof(struct socketSendResources));
  ring->send.transportResources = resources;

  // Create stream for proxy
  CUDACHECK(hipStreamCreateWithFlags(&resources->stream, hipStreamNonBlocking));

  int size = offsetof(struct ncclSendRecvMem, buff)+ring->buffSize;
  CUDACHECK(hipHostAlloc(&resources->hostMem, size, hipHostMallocMapped));
  CUDACHECK(hipHostGetDevicePointer(&resources->devHostMem, resources->hostMem, 0));

  // Just pass the socket info through
  static_assert(sizeof(struct socketInfo) <= sizeof(struct ncclConnect), "socket Connect Info is too big");
  memcpy(connectInfo, myOpaqueInfo, sizeof(struct socketInfo));
  return ncclSuccess;
}

ncclResult_t socketRecvSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct socketRecvResources* resources = (struct socketRecvResources*) malloc(sizeof(struct socketRecvResources));
  ring->recv.transportResources = resources;

  // Create stream for proxy
  CUDACHECK(hipStreamCreateWithFlags(&resources->stream, hipStreamNonBlocking));
  // And event
  for (int i=0; i<MAXSTEPS; i++)
    CUDACHECK(hipEventCreate(resources->syncEvent+i));

  int size = offsetof(struct ncclSendRecvMem, buff)+ring->buffSize;
  CUDACHECK(hipHostAlloc(&resources->hostMem, size, hipHostMallocMapped));
  CUDACHECK(hipHostGetDevicePointer(&resources->devHostMem, resources->hostMem, 0));
  
  // Just pass the socket info through
  struct socketInfo* myInfo = (struct socketInfo*)myOpaqueInfo;
  char ifname[128];
  NCCLCHECK(socketCreateListen(myInfo, ifname));
  resources->listen_fd = myInfo->listen_fd; 
  struct socketInfo* peerInfo = (struct socketInfo*)peerOpaqueInfo;
  INFO("%d -> %d via TCP/%s", peerInfo->rank, myInfo->rank, ifname);
  memcpy(connectInfo, myOpaqueInfo, sizeof(struct socketInfo));
  return ncclSuccess;
}

ncclResult_t socketSendConnect(struct ncclConnect* connectInfo, struct ncclConnector* send) {
  // Setup device pointers
  struct socketSendResources* resources = (struct socketSendResources*)send->transportResources;
  send->conn.buff = resources->devHostMem->buff;
  send->conn.tail = &resources->devHostMem->tail;
  send->conn.opCount = &resources->devHostMem->opCount;
  send->conn.fifo = resources->devHostMem->sizesFifo;

  // Setup receive proxy socket/pointers
  struct socketInfo* info = (struct socketInfo*)connectInfo;
  NCCLCHECK(connectAddress(&info->connect_addr, &resources->fd));
  return ncclSuccess;
}

/* Connect to this peer */
ncclResult_t socketRecvConnect(struct ncclConnect* connectInfo, struct ncclConnector* recv) {
  // Setup device pointers
  struct socketRecvResources* resources = (struct socketRecvResources*)recv->transportResources;
  recv->conn.head = &resources->devHostMem->head;

  // We will finish the socket setup at beginning of Recv proxy
  resources->fd = 0;
  return ncclSuccess;
}

ncclResult_t socketSendFree(void* transportResources) {
  struct socketSendResources* resources = (struct socketSendResources*)transportResources;
  SYSCHECK(close(resources->fd), "close");
  CUDACHECK(hipStreamDestroy(resources->stream));
  CUDACHECK(hipHostFree(resources->hostMem));
  free(resources);
  return ncclSuccess;
}

ncclResult_t socketRecvFree(void* transportResources) {
  struct socketRecvResources* resources = (struct socketRecvResources*)transportResources;
  SYSCHECK(close(resources->listen_fd), "close");
  SYSCHECK(close(resources->fd), "close");
  CUDACHECK(hipStreamDestroy(resources->stream));
  for (int i=0; i<MAXSTEPS; i++) {
    CUDACHECK(hipEventDestroy(resources->syncEvent[i]));
  }
  CUDACHECK(hipHostFree(resources->hostMem));
  free(resources);
  return ncclSuccess;
}

ncclResult_t socketSendProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct socketSendResources* resources = (struct socketSendResources*) (ring->send.transportResources);
  struct ncclSendRecvMem* devMem = ring->devMem;
  volatile int* prevTail = &resources->hostMem->tail;
  int* prevHead = &devMem->head;
  char* localBuff = resources->hostMem->buff;
  int* sizesFifo = resources->hostMem->sizesFifo;
  int buffSize = ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  int head = 0;
  int offset = 0;

  // Update in case we skipped some collectives
  resources->hostMem->opCount = args->opCount;

  while (head < args->nsteps) {
    // Receive from GPU
    transportProxyWait([=] { return head != *prevTail; });

    // Send to socket
    int size = sizesFifo[head%args->substeps];
    NCCLCHECK(socketSend(resources->fd, &size, sizeof(size)));
    NCCLCHECK(socketSend(resources->fd, localBuff+offset, size));
    head++;
    CUDACHECK(hipMemcpyAsync(prevHead, &head, sizeof(int), hipMemcpyHostToDevice, resources->stream));

    offset += sliceSize;
    if (offset == buffSize)
      offset = 0;
  }
  // Ensure all updates are pushed
  CUDACHECK(hipStreamSynchronize(resources->stream));

  // Reset
  *prevTail = 0;
  resources->hostMem->opCount = args->opCount+1;
  return ncclSuccess;
}

ncclResult_t socketRecvProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct socketRecvResources* resources = (struct socketRecvResources*) (ring->recv.transportResources);
  struct ncclSendRecvMem* devMem = ring->devMem;
  int* nextTail = &devMem->tail;
  int* nextOpCount = &devMem->opCount;
  volatile int* nextHead = &resources->hostMem->head;
  char* localBuff = resources->hostMem->buff;
  char* nextBuff = devMem->buff;
  int buffSize = ring->buffSize;
  int sliceSize = buffSize / args->substeps;
  assert(MAXSTEPS >= args->substeps);

  if (resources->fd == 0) {
    struct sockaddr_in sockaddr;
    socklen_t socklen = sizeof(struct sockaddr_in);
    SYSCHECKVAL(accept(resources->listen_fd, (struct sockaddr*)&sockaddr, &socklen), "accept", resources->fd);
  }

  int val = 0;
  while (val != args->opCount) {
    CUDACHECK(hipMemcpyAsync(&val, nextOpCount, sizeof(int), hipMemcpyDeviceToHost, resources->stream));
    CUDACHECK(hipStreamSynchronize(resources->stream));
  }
  int head = 0;
  int offset = 0;

  while (head < args->nsteps) {
    // Receive from socket
    CUDACHECK(hipEventSynchronize(resources->syncEvent[head%args->substeps]));
    int size;
    NCCLCHECK(socketReceive(resources->fd, &size, sizeof(size)));
    NCCLCHECK(socketReceive(resources->fd, localBuff+offset, size));

    // Send to GPU
    transportProxyWait([=] { return (head - *nextHead) < args->substeps; });
    CUDACHECK(hipMemcpyAsync(nextBuff+offset, localBuff+offset, size, hipMemcpyHostToDevice, resources->stream));
    CUDACHECK(hipEventRecord(resources->syncEvent[head%args->substeps], resources->stream));
    head++;
    CUDACHECK(hipMemcpyAsync(nextTail, &head, sizeof(int), hipMemcpyHostToDevice, resources->stream));

    offset += sliceSize;
    if (offset == buffSize)
      offset = 0;
  }
  // Ensure all updates are pushed
  CUDACHECK(hipStreamSynchronize(resources->stream));

  // Wait for last ack and reset
  transportProxyWait([=] { return *nextHead == head; });
  *nextHead = 0;

  return ncclSuccess;
}

struct ncclTransport socketTransport = {
  "TCP",
  socketFillInfo,
  socketCanConnect,
  socketGetRings,
  { socketSendSetup, socketSendConnect, socketSendFree, socketSendProxy },
  { socketRecvSetup, socketRecvConnect, socketRecvFree, socketRecvProxy }
};
