#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2

// !!! Don't change that or the last sync will block
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ T* sharedNextOutput;
  struct ncclComm* comm = args.comm;
  struct ncclRing* ring = comm->rings+bid;
  int prevdirect = ring->recv.conn.direct;
  int nextdirect = ring->send.conn.direct;

  WaitFlag waitDoneFromNext(ring->send.conn.head, NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring->recv.conn.tail, NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring->recv.conn.head, NUM_SUBSTEPS, NULL, 0);
  PostFlag postReadyToNext(ring->send.conn.tail, 0, ring->send.conn.fifo, NUM_BUFCHUNKS*NUM_SUBSTEPS);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const ssize_t size = args.N;
  //const int rank = comm->rank;
  const int nranks = comm->nRanks;
  const int buffSize = ring->buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;

  if (tid == 0) {
    // Update in case we skipped some collectives
    *ring->recv.conn.opCount = args.opCount;
    // Wait for next to be ready
    WaitFlag waitOpCountNext(ring->send.conn.opCount, 0);
    waitOpCountNext.wait(args.opCount);
    if (prevdirect) {
      *ring->recv.conn.ptrExchange = args.ThisOutput;
    }
    if (nextdirect) {
      void* volatile* ptr = &(ring->devMem->ptrExchange);
      while (*ptr == nullptr);
      sharedNextOutput = (T*)*ptr;
      *ptr = nullptr;
    }
  }
  __syncthreads();

  uint64_t step = 0ULL;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput = args.ThisOutput;
  T * __restrict__ prevInput = (T*)ring->recv.conn.buff;
  T * __restrict__ nextOutput = (T*)ring->send.conn.buff;

  for (ssize_t gridOffset = 0; gridOffset < size; gridOffset += gridDim.x*nranks*sliceSize) {
    int chunkSize = min(sliceSize, DIVUP(size-gridOffset,nranks*gridDim.x));
    ALIGN_SIZE(chunkSize, THREADS*sizeof(uint64_t)/sizeof(T));
    ssize_t chunkOffset = gridOffset + bid*nranks*chunkSize;

    /////////////// begin AllReduce steps ///////////////
    ssize_t offset;
    int maxOffset;
    int slice;

    // step 0: push data to next GPU
    slice = ring->devUserRanks[nranks-1];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::Copy(
        thisInput  + offset,
        nextOutput + noffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext,
        postReadyToNext);

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      slice = ring->devUserRanks[nranks-j];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      Prims::Reduce(
          prevInput  + poffset,
          thisInput  + offset,
          nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = ring->devUserRanks[0];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::ReduceCopy(
        prevInput  + poffset,
        thisInput  + offset,
        nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
        thisOutput + offset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP;

    // k-2 steps: copy to next GPU
    if (prevdirect) {
      for (int j=1; j<nranks-1; ++j) {
        slice = ring->devUserRanks[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::Copy(
            thisOutput + offset,
	    nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }
      Prims::Copy(
          NULL,
          NULL,
          0, 0,
          step,
          waitReadyFromPrev,
          postDoneToPrev);
    } else {
      for (int j=1; j<nranks-1; ++j) {
        slice = ring->devUserRanks[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::DoubleCopy(
            prevInput + poffset,
            thisOutput + offset,
	    nextdirect ? (sharedNextOutput + offset) : (nextOutput + noffset),
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }

      // Make final copy from buffer to dest.
      slice = ring->devUserRanks[1];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          prevInput + poffset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitReadyFromPrev,
          postDoneToPrev);
    }
  }

  if (tid == 0) {
    // Wait for next to have consumed all data before we reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step + NUM_BUFCHUNKS));
    *ring->send.conn.head = 0ULL;
    *ring->recv.conn.tail = 0ULL;
    __threadfence_system();
    *ring->recv.conn.opCount = args.opCount+1;
  }
}

#include "ll_kernel.h"

#define NEXT_STEP_LL \
  poffset = noffset; \
  pflag = nflag; \
  noffset += llSliceSize; \
  if (noffset == llBuffSize) { noffset = 0; } \
  nflag++; \
  step++;

template<int THREADS, class FUNC, typename T>
__global__ void AllReduceKernelSmall(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  struct ncclComm* comm = args.comm;
  struct ncclRing* ring = comm->rings+bid;
  volatile uint64_t * recvHeadPtr = ring->recv.conn.llHead;
  volatile uint64_t * sendHeadPtr = ring->send.conn.llHead;
  volatile int * sizesFifo = ring->send.conn.llFifo;
  uint64_t sendHead = sendHeadPtr[0];

  typedef LLPrimitives<THREADS, T, FUNC> LL;

  const ssize_t size = args.N;
  //const int rank = comm->rank;
  const int nranks = comm->nRanks;
  const int llBuffSize = LL_BUFF_SIZE / (2*sizeof(uint64_t));
  const int llSliceSize = llBuffSize / NUM_LL_CHUNKS;
  const int sliceSize = llSliceSize * sizeof(uint64_t) / sizeof(T);

  uint64_t step = ring->send.conn.llStep;
  uint32_t pflag, nflag = step + 1;
  int poffset, noffset = llSliceSize * STEP_TO_SLOT(step);

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput = args.ThisOutput;
  union ncclLLFifoLine * prevInput = (union ncclLLFifoLine *)ring->recv.conn.llBuff;
  union ncclLLFifoLine * nextOutput = (union ncclLLFifoLine *)ring->send.conn.llBuff;

  for (ssize_t chunkOffset = 0; chunkOffset < size; chunkOffset += nranks*sliceSize) {
    int chunkSize = min(sliceSize, DIVUP(size-chunkOffset,nranks));
    ALIGN_SIZE(chunkSize, THREADS*sizeof(uint64_t)/sizeof(T));

    /////////////// begin AllReduce steps ///////////////
    ssize_t offset;
    int maxOffset;
    int slice;

    // step 0: push data to next GPU
    slice = ring->devUserRanks[nranks-1];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    WAIT_NEXT;
    LL::ReduceCopy(
        thisInput  + offset,
        nextOutput + noffset,
        maxOffset, nflag);
    POST_SIZE;

    NEXT_STEP_LL;

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      slice = ring->devUserRanks[nranks-j];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      WAIT_NEXT;
      LL::ReduceCopy(
          thisInput  + offset,
          prevInput  + poffset,
          nextOutput + noffset,
          maxOffset, pflag, nflag);
      POST_SIZE;
      ACK_PREV;

      NEXT_STEP_LL;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = ring->devUserRanks[0];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    WAIT_NEXT;
    LL::ReduceCopy(
        thisInput  + offset,
        prevInput  + poffset,
        thisOutput + offset,
        nextOutput + noffset,
        maxOffset, pflag, nflag);
    POST_SIZE;
    ACK_PREV;

    NEXT_STEP_LL;

    // k-2 steps: copy to next GPU
    for (int j=1; j<nranks-1; ++j) {
      slice = ring->devUserRanks[nranks - j];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      WAIT_NEXT;
      LL::ReduceCopy(
          prevInput + poffset,
          thisOutput + offset,
          nextOutput + noffset,
          maxOffset, pflag, nflag);
      POST_SIZE;
      ACK_PREV;

      NEXT_STEP_LL;
    }

    // Make final copy from buffer to dest.
    slice = ring->devUserRanks[1];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    // Here we need to copy from buffer to this output.
    LL::ReduceCopy(
        prevInput + poffset,
        thisOutput + offset,
        maxOffset, pflag);
    ACK_PREV;
  }

  FIFO_CLEANING_AND_SAVE_STEP(nflag);
}

#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingAllReduce(const void* sendbuff, void* recvbuff,
    const size_t count, ncclComm* comm, hipStream_t stream) {
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    ArgsSetup(sendbuff, recvbuff, 0, count, comm);
    if (count*sizeof(T) <= comm->llThreshold) {
      NCCLCHECK(transportSaveProxies(1, NUM_LL_CHUNKS, (comm->nRanks)*2-2, comm->nRanks, 2*count*sizeof(T), proxyPatternRing, comm, 1));
      SAVE_KERNEL_SMALL(AllReduceKernelSmall, comm, FUNC, T, stream);
    } else {
      NCCLCHECK(transportSaveProxies(NUM_SUBSTEPS, NUM_BUFCHUNKS, (comm->nRanks)*2-2, comm->nRanks, count*sizeof(T), proxyPatternRing, comm, 0));
      SAVE_KERNEL(AllReduceKernel, comm, UNROLL, FUNC, T, stream);
      comm->opCount++;
    }
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class AllReduce {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      size_t count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingAllReduce<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

ncclResult_t ncclAllReduceFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  return enqueue<AllReduce>(sendbuff, recvbuff, count, datatype, op, root, comm, stream);
}

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclAllReduceFunc, "AllReduce", sendbuff, recvbuff, count, datatype,
      op, 0, comm, stream);
}

