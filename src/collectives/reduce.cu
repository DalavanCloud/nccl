#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 4
#define NUM_BUFCHUNKS 2

// Increase Step and boffset for buffer sync
#define NEXT_STEP \
  step++; \
  boffset += sliceSize; \
  if (boffset == buffSize) boffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void ReduceKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  struct ncclComm* comm = args.comm;
  struct ncclRing* ring = comm->rings+bid;

  WaitFlag waitDoneFromNext(ring->send.conn.head, (1-NUM_BUFCHUNKS)*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring->recv.conn.tail, 0);
  PostFlag postDoneToPrev(ring->recv.conn.head, 0, NULL, 0);
  PostFlag postReadyToNext(ring->send.conn.tail, 0, ring->send.conn.fifo, NUM_BUFCHUNKS*NUM_SUBSTEPS);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = comm->nRanks;
  const int buffSize = ring->buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  const int rank = ring->devUserRanks[0];
  const int prevRank = ring->devUserRanks[nranks-1];
  const int root = args.root;

  if (rank != root && tid == 0) {
    // Wait for next to be ready
    WaitFlag waitOpCountNext(ring->send.conn.opCount, 0);
    waitOpCountNext.wait(args.opCount);
  }
  __syncthreads();
  
  int step = 0;
  int boffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput = args.ThisOutput;
  T * __restrict__ prevInput = (T*)ring->recv.conn.buff;
  T * __restrict__ nextOutput = (T*)ring->send.conn.buff;

  for (int gridOffset = 0; gridOffset < size; gridOffset += gridDim.x*sliceSize) {
    int chunkSize = min(sliceSize, DIVUP(size-gridOffset,gridDim.x));
    ALIGN_SIZE(chunkSize, THREADS*UNROLL*sizeof(uint64_t)/sizeof(T));
    int offset = gridOffset + bid*chunkSize;
    int maxOffset = min(chunkSize, size-offset);
    if (prevRank == root) {
      Prims::Copy(
          thisInput + offset,
          nextOutput + boffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext,
          postReadyToNext);
    } else if (rank == root) {
      Prims::Reduce(
          prevInput  + boffset,
          thisInput + offset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitReadyFromPrev,
          postDoneToPrev);
    } else {
      Prims::Reduce(
          prevInput + boffset,
          thisInput + offset,
          nextOutput + boffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);
    }
    NEXT_STEP; // Increases step, boffset
  }

  if (tid == 0) {
    if (rank != root) { 
      // Wait for next to have consumed data before resetting the flag
      waitDoneFromNext.wait(NUM_SUBSTEPS*(step + NUM_BUFCHUNKS - 1));
      *ring->send.conn.head = 0;
    }
    *ring->recv.conn.tail = 0;
    __threadfence_system();
    *ring->recv.conn.opCount = args.opCount+1;
  }
}

#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingReduce(const void* sendbuff, void* recvbuff, const size_t count, const int root,
    ncclComm* comm, hipStream_t stream) {
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportStartProxies(NUM_SUBSTEPS, NUM_BUFCHUNKS, 1, 1, count*sizeof(T), proxyPatternTo(root), comm));
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, root, count, comm);
    LAUNCH_KERNEL(ReduceKernel, comm->nThreads, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template<typename> class RedOp>
class ReduceFunctor {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      size_t count, int root, ncclComm* comm, hipStream_t stream) {
    return RingReduce<RedOp<T>, T>(sendbuff, recvbuff, count, root, comm, stream);
  }
};

ncclResult_t ncclReduceFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  return enqueue<ReduceFunctor>(sendbuff, recvbuff, count, datatype, op, root, comm, stream);
}

NCCL_API(ncclResult_t, ncclReduce, const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclReduce(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  return enqueueCheck(ncclReduceFunc, "Reduce", sendbuff, recvbuff, count, datatype,
      op, root, comm, stream);
}

